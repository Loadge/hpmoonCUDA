#include "hip/hip_runtime.h"
/**
 * @file evaluation.cu
 * @author Miguel Sánchez Tello
 * @date 26/06/2016
 * @brief File with the necessary implementation for the evaluation of the individuals
 *
 */

/********************************** Includes **********************************/

#include "evaluation.h"
#include "hv.h"
#include <stdio.h> // fprintf...
#include <stdlib.h> // malloc, rand...
#include <string.h> // memset...
#include <math.h> // sqrt, INFINITY...

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cub-1.5.2/hipcub/hipcub.hpp>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

/********************************* Methods ********************************/

const int BLOCK_SIZE=32;

/**
 * @brief Given a integer, returns it's next power of two.
 * @param n The initial number.
 * @return Next power of two integer of n.
 */
static inline int nextPowerOfTwo(int n) {
    n--;

    n = n >>  1 | n;
    n = n >>  2 | n;
    n = n >>  4 | n;
    n = n >>  8 | n;
    n = n >> 16 | n;

    return ++n;
}

__global__ static
void cuda_CheckEstructure(	
						bool * mapping
					)
{

}


/**
 * @brief Part of K-means algorithm that is made within the GPU that minimizes the within-cluster sum of squares.
 * @param bigdistCentroids The instances that are considered the center of each cluster. They are formatted to make appropiate calculations.
 * @param NextPowerTotalDistances The size of bigdistCentroids.
 * @param BlockSumWithin Where the results of each concurrent block are stored for further processing in CPU.
 * @param newMapping The mapping the algorithm is going to build in this iteration.
 * @param newMapping The count of instances that each cluster contains.
 */
__global__ static 
void cuda_WithinCluster(
							int * ind,
							int * numWithinBlocks,
							const int * NextPowerTotalDistances,
							const float * __restrict__ bigdistCentroids,
							float * __restrict__ BlockSumWithin
							)
{
	const int gpu_NextPowerTotalDistances = *NextPowerTotalDistances;
	const int gpu_posIndBigDist = *ind * gpu_NextPowerTotalDistances;
	const int gpu_posIndBlockSum = (*ind) * (*numWithinBlocks);
	int idx = blockIdx.x * blockDim.x + threadIdx.x;


	//Specialize BlockReduce for type float
	typedef hipcub::BlockReduce<float, BLOCK_SIZE> BlockReduceT;

	//Allocat temporary storage in shared memory
	__shared__ typename BlockReduceT::TempStorage temp_storage_float;

	float result;
	if(idx < gpu_NextPowerTotalDistances)
		result = BlockReduceT(temp_storage_float).Sum(bigdistCentroids[gpu_posIndBigDist + idx]);
	__syncthreads();
	if(threadIdx.x == 0){
		BlockSumWithin[gpu_posIndBlockSum + blockIdx.x] = result;
	}
}

/**
 * @brief Part of K-means algorithm that is made within the GPU that calculates the euclidean distances between each instance.
 * @param dataBase The database which will contain the instances and the features.
 * @param centroids The instances that are considered the center of each cluster.
 * @param member_chromosome The chromosome necessary to calculate each euclidean distance.
 * @param newMapping The mapping the algorithm is going to build in this iteration.
 * @param newMapping The count of instances that each cluster contains.
 */
__global__ static
void cuda_Convergence_Euclidean(
						int * ind,
						int * tamPoblacion,
						float *dataBase, 
						float *centroids,
						unsigned char *member_chromosomes,
						float * distCentroids,
						bool * newMapping,
						int * samples_in_k
					)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int d_totalDistances = KMEANS * N_INSTANCES;

	const int tamCentroids = KMEANS * N_FEATURES;
	const int tamDistCentroids = KMEANS * N_INSTANCES;
	const int tamMapping = KMEANS * N_INSTANCES;
	const int tamSamples = KMEANS;

	__shared__ int sharedThreadLater[N_INSTANCES];

	int posIndMapping = *ind * tamMapping;
	for(int i=threadIdx.x; i < d_totalDistances; i+= blockDim.x){
		newMapping[posIndMapping + i] = false;
	}
	
	int posIndSamples = *ind * tamSamples;
	for(int i=threadIdx.x; i < KMEANS; i+= blockDim.x){
		samples_in_k[posIndSamples + i] = 0;
	}

	__syncthreads();
/* -- */
	for (int i = idx; i < N_INSTANCES; i += blockDim.x) {
		float minDist = INFINITY;
		int selectCentroid = -1;
		int pos = N_FEATURES * i;

		for (int k = 0; k < KMEANS; ++k) {	//51
			float sum = 0.0f;
/* -- */
			int posIndCentroids = (*ind * tamCentroids)     + (k * N_FEATURES);
			int posIndDistCentr = (*ind * tamDistCentroids) + (k * N_INSTANCES);
			int posIndChromosome = *ind * N_FEATURES;
/* -- * /
//			int posIndCentroids = (2 * tamCentroids)     + (k * N_FEATURES);
//			int posIndDistCentr = (2 * tamDistCentroids) + (k * N_INSTANCES);
//			int posIndChromosome = 2 * N_FEATURES;
/* -- */
			for (int f = 0; f < N_FEATURES; ++f) {
//				if (member_chromosomes[posChromosome + f] & 1) {
					//Multiple accesses to global memory. Better if they were in shared memory. OPT
					//sum += (dataBase[pos + f] - centroids[posIndCentroids + f]) * (dataBase[pos + f] - centroids[posIndCentroids + f]);
					sum += 1;
//				}
			}//f

			float euclidean = sqrt(sum);
			distCentroids[posIndDistCentr + i] = euclidean; //Access to global memory. OPT
			if (euclidean < minDist) {
				minDist = euclidean;
				selectCentroid = k;
			}
		}//k
/* -- */
		sharedThreadLater[i]= selectCentroid;
		newMapping[posIndMapping + (selectCentroid * N_INSTANCES) + i] = true;
		__syncthreads();
	}//i

	//We keep this in GPU to avoid unnecessary memory tranfers
	__syncthreads();
	if(idx==0){
		for(int i=0; i<N_INSTANCES; i++){
			samples_in_k[ posIndSamples + sharedThreadLater[i] ]++;
		}
	}
/* -- */
}

/**
 * @brief K-means algorithm implemented with C-CUDA which minimize the within-cluster and maximize Inter-cluster sum of squares (WCSS and ICSS). Uses CPU and GPU for maximum performance.
 * @param pop Current population.
 * @param begin The first individual to evaluate.
 * @param end The "end-1" position is the last individual to evaluate.
 * @param selInstances The instances chosen as initial centroids.
 * @param dataBase The database which will contain the instances and the features.
 */

void CUDA_kmeans(individual *pop, const int begin, const int end, const int *const selInstances, const float *const dataBase) {

	const int host_tamPoblacion = end - begin;
	const int host_totalDistances = KMEANS * N_INSTANCES;
	const int host_totalCoord = KMEANS * N_FEATURES;
	const int host_nextPowerTotalDistances = nextPowerOfTwo(host_totalDistances);

	const int host_tamCentroids = KMEANS * N_FEATURES;
	const int host_tamDistCentroids = KMEANS * N_INSTANCES;
	const int host_tamMapping = KMEANS * N_INSTANCES;
	const int host_tamSamples = KMEANS;

	//Allocate CPU structures. We use hipHostMalloc to be able to use cuda streams asynchronous memory operations.
	size_t size = host_tamPoblacion * host_totalDistances * sizeof(bool);
	bool *host_mapping;		checkCudaErrors(hipHostMalloc((void **)&host_mapping, size));
	bool *host_newMapping;	checkCudaErrors(hipHostMalloc((void **)&host_newMapping, size));

	size = host_tamPoblacion * KMEANS * N_FEATURES * sizeof(float);
	float * host_centroids;						checkCudaErrors(hipHostMalloc((void **)&host_centroids, size));
	size = host_tamPoblacion * N_FEATURES * sizeof(unsigned char);
	unsigned char * host_member_chromosomes;	checkCudaErrors(hipHostMalloc((void **)&host_member_chromosomes, size));
	
	//No es necesario, borrar
	size = host_tamPoblacion * KMEANS * N_INSTANCES * sizeof(float);
	unsigned char * host_distCentroids;	checkCudaErrors(hipHostMalloc((void **)&host_distCentroids, size));
	
	
	/****************************************Allocate device memory ****************************************/
	//No varía
	float *device_dataBase;
	size_t size_1 = N_INSTANCES * N_FEATURES * sizeof(float);
	checkCudaErrors(hipMalloc((void **)&device_dataBase, size_1));

	//one per stream
	float *device_centroids;	
	size_t size_2 = host_tamPoblacion * KMEANS * N_FEATURES * sizeof(float);
	checkCudaErrors(hipMalloc((void **)&device_centroids, size_2));

	//one per stream
	unsigned char *device_member_chromosomes;	
	size_t size_3 = host_tamPoblacion * N_FEATURES * sizeof(unsigned char);
	checkCudaErrors(hipMalloc((void **)&device_member_chromosomes, size_3));

	//one per stream
	float * device_distCentroids;	
	size_t size_4 = host_tamPoblacion * KMEANS * N_INSTANCES * sizeof(float);
	checkCudaErrors(hipMalloc((void **)&device_distCentroids, size_4));

	//one per stream
	bool * device_mapping;		
	size_t size_5 = host_tamPoblacion * KMEANS * N_INSTANCES * sizeof(bool);
	checkCudaErrors(hipMalloc((void **)&device_mapping, size_5));

	//one per stream
	bool * device_newMapping;
	checkCudaErrors(hipMalloc((void **)&device_newMapping, size_5));		

	//one per stream
	int * device_samples_in_k; 
	size_t size_6 = host_tamPoblacion * KMEANS * sizeof(int);
	checkCudaErrors(hipMalloc((void **)&device_samples_in_k, size_6));

	//all share it
	int * device_NextPowerTotalDistances;
	size_t size_7 = sizeof(int);
	checkCudaErrors(hipMalloc(&device_NextPowerTotalDistances, size_7));

	//one per stream
	float * device_bigdistCentroids;	
	size_t size_9 = host_tamPoblacion * host_nextPowerTotalDistances * sizeof(float);
	checkCudaErrors(hipMalloc((void **)&device_bigdistCentroids, size_9));

	//one per stream
	int * device_ind;
	size_t size_10 = sizeof(int);
	checkCudaErrors(hipMalloc((void **)&device_ind, size_10));

	//one per stream
	int * device_numWithinBlocks;
	checkCudaErrors(hipMalloc((void **)&device_numWithinBlocks, size_10));

	//all share it
	int * device_tamPoblacion;
	size_t size_11 = sizeof(int);
	checkCudaErrors(hipMalloc((void **)&device_tamPoblacion, size_11));

	/****************************************Copy data for all streams ****************************************/

	//database. All streams share it.
	checkCudaErrors(hipMemcpy(device_dataBase, dataBase, size_1, hipMemcpyHostToDevice));

	//mapping (one per kernel)
	for (int i = 0; i < host_tamPoblacion * host_totalDistances; ++i) {
		host_mapping[i] = false;
	}
	checkCudaErrors(hipMemcpy(device_mapping, host_mapping, size_5, hipMemcpyHostToDevice));

	//member_chromosomes
	for(int ind=begin; ind < end; ++ind){
		for(int i=0; i < N_FEATURES; i++){
			int pos = (ind*host_tamPoblacion)+i;
			host_member_chromosomes[pos] = pop[ind].chromosome[i];
		}
	}

/* -CHECK- * /
	//NO HACE FALTA, BORRAR:
	for (int ind = begin; ind < end; ++ind) {
		for(int i=0; i<N_FEATURES; i++){
			printf("\ngpu member_chromosomes[%d]= %u", i, host_member_chromosomes[i]);
		}
		printf("\n----");
	}
/* -CHECK- */

	checkCudaErrors(hipMemcpy(device_member_chromosomes, host_member_chromosomes, size_3, hipMemcpyHostToDevice));

/* -CHECK- * /
	for(int ind=begin; ind < end; ++ind){
		for(int i=0; i < N_FEATURES; i++){
			int pos = (ind*host_tamPoblacion)+i;
			printf("\nmember_chromosomes[%d][%d]= %u, original: %u", ind, i, host_member_chromosomes[pos], pop[ind].chromosome[i]);
		}
		
	}
/* -CHECK- */

	//centroids (one per kernel)
	for(int ind=begin; ind < end; ++ind){
		for (int k = 0; k < KMEANS; ++k) {
			int posDataBase = selInstances[k] * N_FEATURES;
			int posCentr = k * N_FEATURES + (ind * host_totalCoord);

			for (int f = 0; f < N_FEATURES; ++f) {
//				printf("\nEscribiendo %f en la posición %d de host_centroids\n", dataBase[posDataBase + f], posCentr + f);
				if (pop[ind].chromosome[f] & 1) {
					host_centroids[posCentr + f] = dataBase[posDataBase + f];
				}
			}
		}
//		printf("\n-----------");
	}
	checkCudaErrors(hipMemcpy(device_centroids, host_centroids, size_2, hipMemcpyHostToDevice));

	//distCentroids (no es necesario, borrar)
	for(int i=0; i<host_tamPoblacion * host_totalDistances; i++){
		host_distCentroids[i]=0;
	}
	checkCudaErrors(hipMemcpy(device_distCentroids, host_distCentroids, size_2, hipMemcpyHostToDevice));


	//tamPoblacion
	checkCudaErrors(hipMemcpy(device_tamPoblacion, &host_tamPoblacion, size_11, hipMemcpyHostToDevice));

    /* PRUEBAS CON LOS STREAMS * /

    const int num_streams = 16; //Theoric limit in Fermi architecture (compute capability 2.0+)
    hipStream_t streams[num_streams];

    float * data[num_streams];

    for(int i=0; i < num_streams; i++){
        hipStreamCreate(&streams[i]);
        
        hipMalloc(&data[i], N*sizeof(float));

        //launch one worker kernel per stream
        kernel<<< 1, 64, 0, streams[i]>>>(data[i], N);

        //launch a dummy kernel on the default stream
//        kernel<<<1, 1>>>(0, 0);
    }

    hipDeviceReset();

    /* PRUEBAS CON LOS STREAMS */


	//Decide number of blocks and threads for each parallel section.
	hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

	unsigned int numEuclideanThreadsPerBlock = BLOCK_SIZE;
	unsigned int numEuclideanBlocks = ((N_INSTANCES+numEuclideanThreadsPerBlock)-1) / numEuclideanThreadsPerBlock;
	if(numEuclideanBlocks==0){numEuclideanBlocks=1;}

	//In order for CUBLAS to support reduction, numWithinThreadsPerBlock must be a power of two.
	unsigned int numWithinThreadsPerBlock = BLOCK_SIZE;
	unsigned int numWithinBlocks = ((host_totalDistances+host_totalDistances)-1) / BLOCK_SIZE;
	if(numWithinBlocks==0){numWithinBlocks=1;}
	if(numEuclideanBlocks > (_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount)){
		printf("WARNING: Your CUDA hardware has insufficient blocks!.\n");
		numEuclideanBlocks = (_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
	}

	/*	----------------------------------TODO:----------------------------------
			 calculate the amount of global memory necessary for the program to execute. Use this:
			    char msg[256];
    			SPRINTF(msg, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
            			(float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
			    printf("%s", msg);
    	----------------------------------TODO:----------------------------------
/* -- */
	bool gpu_converged = false;
//	float gpu_distCentroids[host_tamPoblacion * KMEANS * N_INSTANCES];
//	bool gpu_mapping[host_tamPoblacion * KMEANS * N_INSTANCES];
//	bool gpu_newMapping[host_tamPoblacion * KMEANS * N_INSTANCES];
//	int gpu_samples_in_k[host_tamPoblacion * KMEANS];
//	float gpu_centroids[host_tamPoblacion * KMEANS * N_FEATURES];

	size = host_tamPoblacion * host_tamCentroids * sizeof(float);
	float * gpu_centroids; checkCudaErrors(hipHostMalloc((void **)&gpu_centroids, size));

	size = host_tamPoblacion * host_tamCentroids * sizeof(float);
	float * results_centroids; checkCudaErrors(hipHostMalloc((void **)&results_centroids, size));

	size = host_tamPoblacion * host_tamDistCentroids * sizeof(float);
	float * results_distCentroids; checkCudaErrors(hipHostMalloc((void **)&results_distCentroids, size));
	
	size = host_tamPoblacion * host_tamSamples * sizeof(int);
	int * results_samples; checkCudaErrors(hipHostMalloc((void **)&results_samples, size));

	size = host_tamPoblacion * host_tamMapping * sizeof(bool);
	bool * results_mapping; checkCudaErrors(hipHostMalloc((void **)&results_mapping, size));

	size = host_tamPoblacion * host_tamMapping * sizeof(bool);
	bool * results_newMapping; checkCudaErrors(hipHostMalloc((void **)&results_newMapping, size));

	//Allocate extra device necessary structures 
	//one per stream
	float * device_BlockSumWithin;
	size_t size_12 = host_tamPoblacion * numWithinBlocks * sizeof(float);
	checkCudaErrors(hipMalloc((void **)&device_BlockSumWithin, size_12));

	//In order for CUBLAS to support reduction, size of the structure used must match the number of threads we are using.
	//numWithinThreadsPerBlock is a power of two.
	size= host_tamPoblacion * host_nextPowerTotalDistances * sizeof(float);
	float * gpu_bigdistCentroids;	checkCudaErrors(hipHostMalloc((void **)&gpu_bigdistCentroids, size));

	float * gpu_BlockSumWithin;	checkCudaErrors(hipHostMalloc((void **)&gpu_BlockSumWithin, size_12));

	size = host_tamPoblacion * host_tamDistCentroids * sizeof(float);
	float * gpu_distCentroids;	checkCudaErrors(hipHostMalloc((void **)&gpu_distCentroids, size));

	size = host_tamPoblacion * host_tamMapping * sizeof(bool);
	bool * gpu_mapping;	checkCudaErrors(hipHostMalloc((void **)&gpu_mapping, size));

	size = host_tamPoblacion * host_tamMapping * sizeof(bool);
	bool * gpu_newMapping;	checkCudaErrors(hipHostMalloc((void **)&gpu_newMapping, size));
	
	//------------------------------

	size = host_tamCentroids * sizeof(float);
	float * gpu_Indcentroids;	checkCudaErrors(hipHostMalloc((void **)&gpu_Indcentroids, size));

	size = host_tamSamples * sizeof(int);
	int * gpu_Indsamples_in_k;	checkCudaErrors(hipHostMalloc((void **)&gpu_Indsamples_in_k, size));

	size = host_tamPoblacion * sizeof(float);
	float * gpu_SumWithin_2;	checkCudaErrors(hipHostMalloc((void **)&gpu_SumWithin_2, size));

	float * gpu_SumInter;	checkCudaErrors(hipHostMalloc((void **)&gpu_SumInter, size));

	bool * popConverged = (bool*) malloc(host_tamPoblacion * sizeof(bool));
	for(int i=0; i<host_tamPoblacion; i++){
		popConverged[i]=false;
	}
	bool allConverged = false;
	int nVueltas=0;
	while(!allConverged){
		printf("\nnVueltas=%d", nVueltas);
		// Evaluate all the individuals
		for (int ind = begin; ind < end; ++ind) {	
														//52 GPU
			//Aquí se corren todos los streams una vez. HACER
			checkCudaErrors(hipMemcpy(device_ind, &ind, sizeof(int), hipMemcpyHostToDevice));
			//Use GPU for the heavy computing part.
			cuda_Convergence_Euclidean <<< numEuclideanBlocks, numEuclideanThreadsPerBlock >>> (
																		device_ind,
																		device_tamPoblacion,
																		device_dataBase, 
																		device_centroids, 
																		device_member_chromosomes, 
																		device_distCentroids, 
																		device_newMapping,
																		device_samples_in_k
																		);
			hipDeviceSynchronize();
		}//ind Euclidean
/* -- */
		checkCudaErrors(hipMemcpy(gpu_distCentroids,device_distCentroids, 	size_4, hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(gpu_newMapping, 	 device_newMapping,		size_5, hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(gpu_mapping, 	 device_mapping, 		size_5, hipMemcpyDeviceToHost));
/* -- */
		for (int ind = begin; ind < end; ++ind) {
			int posIndCentroids =  ind * host_tamCentroids;
			int posIndDistCentr =  ind * host_tamDistCentroids;
			int posIndChromosome = ind * N_FEATURES;
			int posIndMapping = ind * host_tamMapping;
			int posIndSamples = ind * host_tamSamples;
/* -- */

			//Has the algorithm converged? We use CPU.
			popConverged[ind] = true;
			int sum=0;
			for(int i=0; i<host_totalDistances; i++){
				sum+= (gpu_mapping[posIndMapping + i] != gpu_newMapping[posIndMapping + i]) ? 0 : 1;
			}
			if(sum != host_totalDistances ){
						popConverged[ind] = false;
			}

			/* -- * /
			for(int i=0; i<host_tamPoblacion; i++){
				printf("\n gpu popConverged [%d]=%d", i, popConverged[i]);
			}
			printf("\n--------");
			/* -- */

/* -- */
			if (!popConverged[ind]) {
				nVueltas++;
				size = host_tamCentroids * sizeof(float);
				checkCudaErrors(hipMemcpy(gpu_Indcentroids, 	device_centroids, 						size, hipMemcpyDeviceToHost));
				
				size = host_tamSamples * sizeof(int);
				checkCudaErrors(hipMemcpy(gpu_Indsamples_in_k, device_samples_in_k + posIndSamples, 	size, hipMemcpyDeviceToHost));

				// Update the position of the centroids
				for (int k = 0; k < KMEANS; ++k) {
					int posCentroids = k * N_FEATURES;
					int posMap = k * N_INSTANCES;
					for (int f = 0; f < N_FEATURES; ++f) {
						float sum = 0.0f;
						if (pop[ind].chromosome[f] & 1) {
							for (int i = 0; i < N_INSTANCES; ++i) {
								if (gpu_newMapping[posIndMapping + posMap + i]) {
									sum += dataBase[(N_FEATURES * i) + f];
								}
							}
							if(gpu_Indsamples_in_k[posIndSamples + k] == 0){
								gpu_Indcentroids[posIndCentroids + posCentroids + f] = 0;
							}else{
								gpu_Indcentroids[posIndCentroids + posCentroids + f] = sum / gpu_Indsamples_in_k[posIndSamples + k];
							}
						}//if chromosome
					}//for nfeatures
				}//for KMEANS
/* -- */
				//New centroids, thanks to CPU work
				size = host_tamCentroids * sizeof(float);
				checkCudaErrors(hipMemcpy(device_centroids + posIndCentroids, 	gpu_Indcentroids, size, 	hipMemcpyHostToDevice));
/* -- */
			}//!converged
/* -- */
			//Anyone left to converge?
			allConverged = true;
			for(int i=0; i<host_tamPoblacion && allConverged; i++){
				if(!popConverged[i]){
					allConverged = false;
				}
			}
/* -- */
		}//ind Converged
		// Swap GPU mapping tables 
		checkCudaErrors(hipMemcpy(device_mapping, 		gpu_newMapping, size_5, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(device_newMapping, 	gpu_mapping, size_5, 	hipMemcpyHostToDevice));
/* -- */		
	}//while ninguno sin converger		//52
/* -- */

		/************ Minimize the within-cluster and maximize Inter-cluster sum of squares (WCSS and ICSS) *************/
/* -- */
	for (int ind = begin; ind < end; ++ind) {
		int posIndCentroids =  ind * host_tamCentroids;
		int posIndDistCentr =  ind * host_tamDistCentroids;
		int posIndChromosome = ind * N_FEATURES;
		int posIndMapping = ind * host_tamMapping;
		int posIndSamples = ind * host_tamSamples;

		int posIndBigDist = ind * host_nextPowerTotalDistances;
		int k=0;
/* -- */
		for (; k < host_totalDistances; ++k) { 	//50  GPU
//				printf("\nSe va a sumar %f", gpu_distCentroids[posIndDistCentr + k] * gpu_mapping[posIndMapping + k]); 
				gpu_bigdistCentroids[posIndBigDist + k] = gpu_distCentroids[posIndDistCentr + k] * gpu_mapping[posIndMapping + k];			
//				gpu_bigdistCentroids[posIndBigDist + k] = 1.0f;
		}
/* -- */
		for(; k < host_nextPowerTotalDistances; k++){
//			gpu_bigdistCentroids[k] = 0.0f;
			gpu_bigdistCentroids[posIndBigDist + k] = 0.0f;
		}
		/* -CHECK-*/
	}//ind rellenar bigdist

//		printf("\nSe va a mandar a la GPU:");
		for(int k=0; k < host_tamPoblacion * host_nextPowerTotalDistances; k++){
//			printf("\n gpu_bigdistCentroids[%d]=%f", k, gpu_bigdistCentroids[k]);
		}
		/* -CHECK-*/

	checkCudaErrors(hipMemcpy(device_bigdistCentroids,	gpu_bigdistCentroids, 	size_9, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(device_NextPowerTotalDistances, &host_nextPowerTotalDistances, sizeof(int), hipMemcpyHostToDevice));
/* -- */	
	for (int ind = begin; ind < end; ++ind) {
		checkCudaErrors(hipMemcpy(device_ind, &ind, sizeof(int), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(device_numWithinBlocks, &numWithinBlocks, sizeof(int), hipMemcpyHostToDevice));
		// Within-cluster
		cuda_WithinCluster <<< numWithinBlocks, numWithinThreadsPerBlock>>> (
														device_ind,
														device_numWithinBlocks,
														device_NextPowerTotalDistances, 
														device_bigdistCentroids,
														device_BlockSumWithin
														);
		hipDeviceSynchronize();
	}//ind

	checkCudaErrors(hipMemcpy(gpu_BlockSumWithin, 	device_BlockSumWithin, 	size_12, hipMemcpyDeviceToHost));
	for (int ind = begin; ind < end; ++ind) {
		int posIndBlockSum = ind * numWithinBlocks;
		
		for(int i=0; i < numWithinBlocks; i++){
//			printf("\n  sumando %f", gpu_BlockSumWithin[posIndBlockSum + i]);
			gpu_SumWithin_2[ind] += gpu_BlockSumWithin[posIndBlockSum + i];
		}
	}//ind
	
/* -- */
	// Inter-cluster  //TODO: paralelize this if it's worth the effort
	checkCudaErrors(hipMemcpy(gpu_centroids, 		device_centroids,		size_2,  hipMemcpyDeviceToHost));
	for (int ind = begin; ind < end; ++ind) {
		int posIndCentroids =  ind * host_tamCentroids;
		
		for (int k = 0; k < KMEANS; ++k) {
			int gpu_posCentroids = k * N_FEATURES;
			for (int i = gpu_posCentroids + N_FEATURES; i < host_totalCoord; i += N_FEATURES) {
				float sum = 0.0f;
				for (int f = 0; f < N_FEATURES; ++f) {
					if (pop[ind].chromosome[f] & 1) {
						sum += (gpu_centroids[posIndCentroids + gpu_posCentroids + f] 
							-   gpu_centroids[posIndCentroids + i + f]) 
							*  (gpu_centroids[posIndCentroids + gpu_posCentroids + f] 
							-   gpu_centroids[posIndCentroids + i + f]);
					}
				}
				gpu_SumInter[ind] += sqrt(sum);
			}
		}
	}//ind
	for (int ind = begin; ind < end; ++ind) {
		// First objective function (Within-cluster sum of squares (WCSS))
		pop[ind].fitness[0] = gpu_SumWithin_2[ind];

		// Second objective function (Inter-cluster sum of squares (ICSS))
		pop[ind].fitness[1] = gpu_SumInter[ind];


		printf("\ngpu sumWithin[%d]=%f", ind, gpu_SumWithin_2[ind]);
		printf("\ngpu sumInter[%d]=%f", ind, gpu_SumInter[ind]);

		// Third objective function (Number of selected features)
		//pop[ind].fitness[2] = (float) nSelFeatures;

//		checkCudaErrors(hipFree(d_BlockSumWithin));
		//WCSS and ICSS minimization process
/* -- */
	}//ind
/* -- */
	checkCudaErrors(hipMemcpy(results_centroids,		device_centroids, 		size_2, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(results_distCentroids,	device_distCentroids, 	size_4, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(results_samples,			device_samples_in_k, 	size_6, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(results_mapping,			device_mapping, 		size_5, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(results_newMapping,		device_newMapping, 		size_5, hipMemcpyDeviceToHost));

/* -CHECK- */			//CHECKS
	float auxSumaCentroids=0;
	float auxSumaDist=0;
	int auxSumaSamples=0;
	int auxSumaMapping=0;
	int auxSumaNewMapping=0;
	for(int i=0; i<host_tamPoblacion * KMEANS*N_FEATURES; i++){
		auxSumaCentroids += results_centroids[i];
//		printf("\nGPU results_centroids[%d]= %f", i, results_centroids[i]);
	}
	for(int i=0; i<host_tamPoblacion * KMEANS*N_INSTANCES; i++){
		auxSumaDist += results_distCentroids[i];
//			printf("\nGPU results_distCentroids[%d]= %f", i, results_distCentroids[i]);
	}	
	for(int i=0; i<host_tamPoblacion * host_tamSamples; i++){
		auxSumaSamples += results_samples[i];
//		printf("\nGPU samples_in_k[%d]=%d", i, results_samples[i]);
	}
	for(int i=0; i<host_tamPoblacion * host_tamMapping; i++){
		auxSumaMapping += results_mapping[i];
//		printf("\nGPU mapping[%d]=%d", i, results_mapping[i]);
	}
	for(int i=0; i<host_tamPoblacion * host_tamMapping; i++){
		auxSumaNewMapping += results_newMapping[i];
//		printf("\nGPU newMapping[%d]=%d", i, results_newMapping[i]);
	}

	printf("\nnVueltas=%d", nVueltas);

	printf("\nGPU: Suma total de results_centroids: %f", auxSumaCentroids);
	printf("\nGPU: Suma total de results_distCentroids: %f", auxSumaDist);
	printf("\nGPU: Suma total de results_samples: %d", auxSumaSamples);
	printf("\nGPU: Suma total de newMapping: %d", auxSumaMapping);
	printf("\nGPU: Suma total de newMapping: %d", auxSumaNewMapping);
	printf("\nvalor de host_nextPowerTotalDistances= %d", host_nextPowerTotalDistances);
/* -CHECK- */

/* -- * /
	// Resources used are released
	checkCudaErrors(hipFree(device_dataBase));
	checkCudaErrors(hipFree(device_centroids));
	checkCudaErrors(hipFree(device_member_chromosomes));
	checkCudaErrors(hipFree(device_distCentroids));
	checkCudaErrors(hipFree(device_mapping));
	checkCudaErrors(hipFree(device_newMapping));
	checkCudaErrors(hipFree(device_samples_in_k));
	checkCudaErrors(hipFree(device_posCentroids));
	checkCudaErrors(hipFree(device_posDistCentr));
	checkCudaErrors(hipFree(device_NextPowerTotalDistances));
	checkCudaErrors(hipFree(device_bigmapping));
	checkCudaErrors(hipFree(device_bigdistCentroids));
/* -- * /
	free(mapping);
	free(newMapping);
/* -- */

}//CUDA_kmeans

/**
 * @brief Sequential K-means algorithm which minimize the within-cluster and maximize Inter-cluster sum of squares (WCSS and ICSS).
 * @param pop Current population.
 * @param begin The first individual to evaluate.
 * @param end The "end-1" position is the last individual to evaluate.
 * @param selInstances The instances choosen as initial centroids.
 * @param dataBase The database which will contain the instances and the features.
 */

void test_cpu_kmeans(individual *pop, const int begin, const int end, const int *const selInstances, const float *const dataBase) {
	const int host_tamPoblacion = end - begin;
	const int host_totalDistances = KMEANS * N_INSTANCES;
	const int host_totalCoord = KMEANS * N_FEATURES;
	const int host_nextPowerTotalDistances = nextPowerOfTwo(host_totalDistances);

	const int host_tamCentroids = KMEANS * N_FEATURES;
	const int host_tamDistCentroids = KMEANS * N_INSTANCES;
	const int host_tamMapping = KMEANS * N_INSTANCES;
	const int host_tamSamples = KMEANS;
	const int totalCoord = KMEANS * N_FEATURES;

	bool *mapping = (bool*) malloc(host_tamPoblacion * host_tamMapping * sizeof(bool));
	bool *newMapping = (bool*) malloc(host_tamPoblacion * host_tamMapping * sizeof(bool));

	float centroids[host_tamPoblacion * host_tamCentroids];

	// The centroids will have the selected features of each individual
	for(int ind=begin; ind < end; ++ind){
		for (int k = 0; k < KMEANS; ++k) {
			int posDataBase = selInstances[k] * N_FEATURES;
			int posCentr = k * N_FEATURES + (ind * host_totalCoord);

			for (int f = 0; f < N_FEATURES; ++f) {
//				printf("\nEscribiendo %f en la posición %d de host_centroids\n", dataBase[posDataBase + f], posCentr + f);
				if (pop[ind].chromosome[f] & 1) {
					centroids[posCentr + f] = dataBase[posDataBase + f];
				}
			}
		}
//		printf("\n-----------");
	}

	// Initialize the array of minimum distances and the mapping table
	const int totalDist = KMEANS * N_INSTANCES;
	float distCentroids[host_tamPoblacion * KMEANS * N_INSTANCES];
	int samples_in_k[host_tamPoblacion * KMEANS];

	// Initialize the mapping table
	for (int i = 0; i < host_tamPoblacion * host_totalDistances; ++i) {
		mapping[i] = false;
	}
	for(int i=0; i<host_tamPoblacion * host_totalDistances; i++){
		distCentroids[i]=0;
	}

/* -CHECK- * /
	//NO HACE FALTA, BORRAR:
	for (int ind = begin; ind < end; ++ind) {
		for(int i=0; i<N_FEATURES; i++){
//			printf("\ntest member_chromosomes[%d]= %u", i, pop[ind].chromosome[i]);
		}
		printf("\n----");
	}
/* -CHECK- */

	// Evaluate all the individuals
//	for (int ind = begin; ind < end; ++ind) {
	bool * popConverged = (bool*) malloc(host_tamPoblacion * sizeof(bool));
	for(int i=0; i<host_tamPoblacion; i++){
		popConverged[i]=false;
	}
	bool allConverged = false;

	int nVueltas=0;
	while(!allConverged){
		printf("\nnVueltas=%d", nVueltas);

		/******************** Convergence process *********************/
		
//		for (int maxIter = 0; maxIter < MAX_ITER_KMEANS && !converged; ++maxIter) {	//52 test
		for (int ind = begin; ind < end; ++ind) {
			
			// The mapping table is cleaned in each iteration
			int posIndMapping = ind * host_tamMapping;
			for (int i = 0; i < totalDist; ++i) {
				newMapping[posIndMapping + i] = false;
			}
			
			int posIndSamples = ind * host_tamSamples;
			for (int i = 0; i < KMEANS; ++i) {
				samples_in_k[posIndSamples + i] = 0;
			}

			// Calculate all distances (Euclidean distance) between each instance and the centroids
			for (int i = 0; i < N_INSTANCES; ++i) {
				float minDist = INFINITY;
				int selectCentroid = -1;
				int pos = N_FEATURES * i;
				posIndMapping = (ind * host_tamMapping);

				for (int k = 0; k < KMEANS; ++k) {
					float sum = 0.0f;
					int posIndCentroids = (ind * host_tamCentroids)     + (k * N_FEATURES);
					int posIndDistCentr = (ind * host_tamDistCentroids) + (k * N_INSTANCES);
					int posIndChromosome = ind * N_FEATURES;
					for (int f = 0; f < N_FEATURES; ++f) {
//						if (pop[ind].chromosome[f] & 1) {
							//sum += (dataBase[pos + f] - centroids[posIndCentroids + f]) * (dataBase[pos + f] - centroids[posIndCentroids + f]);
							sum += 1;
//						}
					}

					float euclidean = sqrt(sum);
					distCentroids[posIndDistCentr + i] = euclidean;
					if (euclidean < minDist) {
						minDist = euclidean;
						selectCentroid = k;
					}
				}

				newMapping[posIndMapping + (selectCentroid * N_INSTANCES) + i] = true;
				samples_in_k[posIndSamples + selectCentroid]++;
			}
		}//ind Euclidean
/* -- */
		for (int ind = begin; ind < end; ++ind) {
/* -- */
			int posIndCentroids =  ind * host_tamCentroids;
			int posIndDistCentr =  ind * host_tamDistCentroids;
			int posIndChromosome = ind * N_FEATURES;
			int posIndMapping = ind * host_tamMapping;
			int posIndSamples = ind * host_tamSamples;

/* -ALTERNATIVA 1- * /	// Has the algorithm converged for every individual?
			popConverged[ind] = true;
			for (int k = 0; k < KMEANS && popConverged[ind]; ++k) {
				int posMap = k * N_INSTANCES;
				for (int i = 0; i < N_INSTANCES && popConverged[ind]; ++i) {
					printf("\nViendo si newMapping[%d]=%d y mapping[%d]=%d son iguales o no",  posIndMapping + posMap + i,
																							   newMapping[posIndMapping + posMap + i],
																							   posIndMapping + posMap + i,
																							   mapping[posIndMapping + posMap + i]);
					if (newMapping[posIndMapping + posMap + i] != mapping[posIndMapping + posMap + i]) {
						popConverged[ind] = false;
					}
				}
			}
/* -ALTERNATIVA 2- */
			//Has the algorithm converged? We use CPU.
			popConverged[ind] = true;
			int sum=0;
			for(int i=0; i<host_totalDistances; i++){
				sum+= (mapping[posIndMapping + i] != newMapping[posIndMapping + i]) ? 0 : 1;
			}
			if(sum != host_totalDistances ){
						popConverged[ind] = false;
			}
			/* -- * /
			for(int i=0; i<host_tamPoblacion; i++){
				printf("\n test popConverged [%d]=%d", i, popConverged[i]);
			}
			printf("\n--------");		
			/* -- */
/* -- */
			if (!popConverged[ind]) {
				nVueltas++;

				// Update the position of the centroids
				for (int k = 0; k < KMEANS; ++k) {
					int posCentr = k * N_FEATURES;
					int posMap = k * N_INSTANCES;
					for (int f = 0; f < N_FEATURES; ++f) {
						float sum = 0.0f;
						if (pop[ind].chromosome[f] & 1) {
							for (int i = 0; i < N_INSTANCES; ++i) {
								if (newMapping[posIndMapping + posMap + i]) {
									sum += dataBase[(N_FEATURES * i) + f];
								}
							}
							if(samples_in_k[posIndSamples + k] == 0){
								centroids[(ind * host_tamCentroids) + posCentr + f] = 0;
							}else{
								centroids[(ind * host_tamCentroids) + posCentr + f] = sum / samples_in_k[posIndSamples + k];
							}
						}
					}
				}
/* -- */
			}
/* -- */
			//Anyone left to converge?
			allConverged = true;
			for(int i=0; i<host_tamPoblacion && allConverged; i++){
				if(!popConverged[i]){
					allConverged = false;
				}
			}
/* -- */
		}//ind Converged
		// Swap mapping tables
		bool *aux = newMapping;
		newMapping = mapping;
		mapping = aux;
/* -- */
	}//ninguno sin converger		//52

		/************ Minimize the within-cluster and maximize Inter-cluster sum of squares (WCSS and ICSS) *************/

	float sumWithin[host_tamPoblacion];
	float sumInter[host_tamPoblacion];
	
	for (int ind = begin; ind < end; ++ind) {
			int posIndCentroids =  ind * host_tamCentroids;
			int posIndDistCentr =  ind * host_tamDistCentroids;
			int posIndChromosome = ind * N_FEATURES;
			int posIndMapping = ind * host_tamMapping;
			int posIndSamples = ind * host_tamSamples;

		for (int k=0; k < host_totalDistances; ++k) { 	//50  test
//				printf("\nSe va a sumar %f", gpu_distCentroids[posIndDistCentr + k] * gpu_mapping[posIndMapping + k]); 
				sumWithin[ind] += distCentroids[posIndDistCentr + k] * mapping[posIndMapping + k];			
//				sumWithin[ind] += distCentroids[posIndDistCentr + k] = 1.0f;
		}
	}//ind
	for (int ind = begin; ind < end; ++ind) {
		int posIndCentroids =  ind * host_tamCentroids;
		for (int k = 0; k < KMEANS; ++k) {
			int posCentr = (k * N_FEATURES);
			// Inter-cluster
			for (int i = posCentr + N_FEATURES; i < totalCoord; i += N_FEATURES) {
				float sum = 0.0f;
				for (int f = 0; f < N_FEATURES; ++f) {
					if (pop[ind].chromosome[f] & 1) {
						sum += (centroids[posIndCentroids + posCentr + f]
						    -  centroids[i + f])
						    * (centroids[posIndCentroids +  posCentr + f]
						    -  centroids[i + f]);
					}
				}
				sumInter[ind] += sqrt(sum);
			}
		}
	}//for each individual

	for (int ind = begin; ind < end; ++ind) {

		// First objective function (Within-cluster sum of squares (WCSS))
		pop[ind].fitness[0] = sumWithin[ind];

		// Second objective function (Inter-cluster sum of squares (ICSS))
		pop[ind].fitness[1] = sumInter[ind];

		printf("\ntest sumWithin[%d]=%f", ind, sumWithin[ind]);
		printf("\ntest sumInter[%d]=%f", ind, sumInter[ind]);

		// Third objective function (Number of selected features)
		//pop[ind].fitness[2] = (float) nSelFeatures;
	}//ind

	
/* -- */
	float auxSumaCentroids=0;			//CHECKS
	float auxSumaDist=0;
	int auxSumaSamples=0;
	int auxSumaMapping=0;
	int auxSumaNewMapping=0;
	for(int i=0; i<host_tamPoblacion * KMEANS*N_FEATURES; i++){
		auxSumaCentroids += centroids[i];
//		printf("\ntest centroids[%d]= %f", i, centroids[i]);
	}
	
	//Euclidean
	for(int i=0; i<host_tamPoblacion * KMEANS*N_INSTANCES; i++){
		auxSumaDist += distCentroids[i];
//		printf("\ntest test_distCentroids[%d]= %f", i, distCentroids[i]);
	}
	for(int i=0; i<host_tamPoblacion * KMEANS; i++){
		auxSumaSamples += samples_in_k[i];
//		printf("\ntest samples_in_k[%d]=%d", i, samples_in_k[i]);
	}
	for(int i=0; i<host_tamPoblacion * host_tamMapping; i++){
		auxSumaMapping += mapping[i];
//		printf("\ntest mapping[%d]=%d", i, mapping[i]);
	}
	for(int i=0; i<host_tamPoblacion * host_tamMapping; i++){
		auxSumaNewMapping += newMapping[i];
//		printf("\ntest newMapping[%d]=%d", i, newMapping[i]);
	}

	printf("\nnVueltas=%d", nVueltas);
	//(WCSS and ICSS)
	
		

	printf("\nTEST: Suma total de centroids: %f", auxSumaCentroids);
	printf("\nTEST: Suma total de distCentroids: %f", auxSumaDist);
	printf("\nTEST: Suma total de samples: %d", auxSumaSamples);
	printf("\nTEST: Suma total de mapping: %d", auxSumaMapping);
	printf("\nTEST: Suma total de newMapping: %d", auxSumaNewMapping);



	// Resources used are released
//	free(mapping);
//	free(newMapping);
/* -- */
}

/**
 * @brief Sequential K-means algorithm which minimize the within-cluster and maximize Inter-cluster sum of squares (WCSS and ICSS).
 * @param pop Current population.
 * @param begin The first individual to evaluate.
 * @param end The "end-1" position is the last individual to evaluate.
 * @param selInstances The instances choosen as initial centroids.
 * @param dataBase The database which will contain the instances and the features.
 */
void cpu_kmeans(individual *pop, const int begin, const int end, const int *const selInstances, const float *const dataBase) {
	const int host_tamPoblacion = end - begin;
	const int host_totalDistances = KMEANS * N_INSTANCES;
	const int host_totalCoord = KMEANS * N_FEATURES;
	const int host_nextPowerTotalDistances = nextPowerOfTwo(host_totalDistances);

	const int host_tamCentroids = KMEANS * N_FEATURES;
	const int host_tamDistCentroids = KMEANS * N_INSTANCES;
	const int host_tamMapping = KMEANS * N_INSTANCES;
	const int host_tamSamples = KMEANS;
	const int totalCoord = KMEANS * N_FEATURES;

	bool *mapping = (bool*) malloc(KMEANS * N_INSTANCES * sizeof(bool));
	bool *newMapping = (bool*) malloc(KMEANS * N_INSTANCES * sizeof(bool));
	const int tamPoblacion = end - begin;

	// Evaluate all the individuals
	int nVueltas=0;
	for (int ind = begin; ind < end; ++ind) {
		const int totalCoord = KMEANS * N_FEATURES;
		float centroids[KMEANS * N_FEATURES];

		// The centroids will have the selected features of the individual
		for (int k = 0; k < KMEANS; ++k) {
			int posDataBase = selInstances[k] * N_FEATURES;
			int posCentr = k * N_FEATURES;

			for (int f = 0; f < N_FEATURES; ++f) {
				if (pop[ind].chromosome[f] & 1) {
					centroids[posCentr + f] = dataBase[posDataBase + f];
				}
			}
		}

		/******************** Convergence process *********************/

		// Initialize the array of minimum distances and the mapping table
		const int totalDist = KMEANS * N_INSTANCES;
		float distCentroids[KMEANS * N_INSTANCES];
		int samples_in_k[KMEANS];

		// Initialize the mapping table
		for (int i = 0; i < totalDist; ++i) {
			mapping[i] = false;
		}

		// To avoid poor performance, at most "MAX_ITER_KMEANS" iterations are executed
		bool converged = false;
		for (int maxIter = 0; maxIter < MAX_ITER_KMEANS && !converged; ++maxIter) {	//52 cpu

			// The mapping table is cleaned in each iteration
			for (int i = 0; i < totalDist; ++i) {
				newMapping[i] = false;
			}
			for (int i = 0; i < KMEANS; ++i) {
				samples_in_k[i] = 0;
			}

			// Calculate all distances (Euclidean distance) between each instance and the centroids
			for (int i = 0; i < N_INSTANCES; ++i) {
				float minDist = INFINITY;
				int selectCentroid = -1;
				int pos = N_FEATURES * i;
				for (int k = 0; k < KMEANS; ++k) {
					float sum = 0.0f;
					int posCentr = k * N_FEATURES;
					int posDistCentr = k * N_INSTANCES;
					for (int f = 0; f < N_FEATURES; ++f) {
						if (pop[ind].chromosome[f] & 1) {
//							sum += (dataBase[pos + f] - centroids[posCentr + f]) * (dataBase[pos + f] - centroids[posCentr + f]);
							sum +=1;
						}
					}
					

					float euclidean = sqrt(sum);
					distCentroids[posDistCentr + i] = euclidean;
					if (euclidean < minDist) {
						minDist = euclidean;
						selectCentroid = k;
					}
				}

				newMapping[(selectCentroid * N_INSTANCES) + i] = true;
				samples_in_k[selectCentroid]++;
			}
/* -- */	
			// Has the algorithm converged?
			converged = true;
			for (int k = 0; k < KMEANS && converged; ++k) { 
				int posMap = k * N_INSTANCES;
				for (int i = 0; i < N_INSTANCES && converged; ++i) {
					if (newMapping[posMap + i] != mapping[posMap + i]) {
						converged = false;
					}
				}	
			}

			if (!converged) {
				nVueltas++;
				// Update the position of the centroids
				for (int k = 0; k < KMEANS; ++k) {
					int posCentr = k * N_FEATURES;
					int posMap = k * N_INSTANCES;
					for (int f = 0; f < N_FEATURES; ++f) {
						float sum = 0.0f;
						if (pop[ind].chromosome[f] & 1) {
							for (int i = 0; i < N_INSTANCES; ++i) {
								if (newMapping[posMap + i]) {
									sum += dataBase[(N_FEATURES * i) + f];
								}
							}

							if(samples_in_k[k] == 0){
								centroids[posCentr + f] = 0;
							}else{
								centroids[posCentr + f] = sum / samples_in_k[k];
							}
						}
					}
				}

				// Swap mapping tables
				bool *aux = newMapping;
				newMapping = mapping;
				mapping = aux;
			}
/* -- */
	}//maxIter 	//52

		/************ Minimize the within-cluster and maximize Inter-cluster sum of squares (WCSS and ICSS) *************/

		float sumWithin = 0.0f;
		float sumInter = 0.0f;
		for (int k = 0; k < KMEANS; ++k) {
			int posCentr = k * N_FEATURES;
			int posDistCentr = k * N_INSTANCES;

			// Within-cluster
			for (int i = 0; i < N_INSTANCES; ++i) {
				if (mapping[posDistCentr + i]) {
					sumWithin += distCentroids[posDistCentr + i];
				}
			}

			// Inter-cluster
			for (int i = posCentr + N_FEATURES; i < totalCoord; i += N_FEATURES) {
				float sum = 0.0f;
				for (int f = 0; f < N_FEATURES; ++f) {
					if (pop[ind].chromosome[f] & 1) {
						sum += (centroids[posCentr + f] - centroids[i + f]) * (centroids[posCentr + f] - centroids[i + f]);
					}
				}
				sumInter += sqrt(sum);
			}
		}

		// First objective function (Within-cluster sum of squares (WCSS))
		pop[ind].fitness[0] = sumWithin;

		// Second objective function (Inter-cluster sum of squares (ICSS))
		pop[ind].fitness[1] = sumInter;

		printf("\ncpu sumWithin=%f", sumWithin);
		printf("\ncpu sumInter=%f", sumInter);

		// Third objective function (Number of selected features)
		//pop[ind].fitness[2] = (float) nSelFeatures;
/* -- */
					//CHECKS
		float auxSumaCentroids=0;	
		float auxSumaDist=0;
		int auxSumaSamples=0;
		int auxSumaMapping=0;
		int auxSumaNewMapping=0;

		for(int i=0; i<KMEANS*N_FEATURES; i++){
			auxSumaCentroids += centroids[i];
//			printf("\ncpu centroids[%d]= %f", i, centroids[i]);
		}
		for(int i=0; i<KMEANS*N_INSTANCES; i++){
			auxSumaDist += distCentroids[i];
//			printf("\ncpu distCentroids[%d]= %f", i, distCentroids[i]);
		}
		for(int i=0; i<KMEANS; i++){
			auxSumaSamples += samples_in_k[i];
//			printf("\ncpu samples_in_k[%d]= %d", i, samples_in_k[i]);
		}
		for(int i=0; i<host_tamMapping; i++){
			auxSumaMapping += mapping[i];
//			printf("\ncpu mapping[%d]=%d", i, mapping[i]);
		}
		for(int i=0; i<host_tamMapping; i++){
			auxSumaNewMapping += newMapping[i];
//			printf("\ncpu newMapping[%d]=%d", i, newMapping[i]);
		}

		printf("\nnVueltas=%d", nVueltas);


		printf("\nCPU: Suma total de centroids: %f", auxSumaCentroids);
		printf("\nCPU: Suma total de distCentroids: %f", auxSumaDist);
		printf("\nCPU: Suma total de samples_in_k: %d", auxSumaSamples);
		printf("\nCPU: Suma total de mapping: %d", 		auxSumaMapping);
		printf("\nCPU: Suma total de newMapping: %d", 	auxSumaNewMapping);
	}//for each individual
	// Resources used are released
	free(mapping);
	free(newMapping);
/* -- */
}

/**
 * @brief Evaluation of each individual.
 * @param pop Current population.
 * @param begin The first individual to evaluate.
 * @param end The "end-1" position is the last individual to evaluate.
 * @param dataBase The database which will contain the instances and the features.
 * @param nInstances The number of instances (rows) of the database.
 * @param nFeatures The number of features (columns) of the database.
 * @param nObjectives The number of objectives.
 * @param selInstances The instances chosen as initial centroids.
 */
void test_cpu_evaluation(individual *pop, const int begin, const int end, const float *const dataBase, const int nInstances, const int nFeatures, const unsigned char nObjectives, const int *const selInstances) {


	/************ Kmeans algorithm ***********/

	// Evaluate all the individuals and get the first and second objective for them
	test_cpu_kmeans(pop, begin, end, selInstances, dataBase);


	/******************** Fitness normalization *********************/

	int totalInd = end - begin;
	for (unsigned char obj = 0; obj < nObjectives; ++obj) {

		// Fitness vector average
		float average = 0;
		for (int i = begin; i < end; ++i) {
			average += pop[i].fitness[obj];
		}

		average /= totalInd;

		// Fitness vector variance
		float variance = 0;
		for (int i = begin; i < end; ++i) {
			variance += (pop[i].fitness[obj] - average) * (pop[i].fitness[obj] - average);
		}
		variance /= (totalInd - 1);

		// Fitness vector standard deviation
		float std_deviation = sqrt(variance);

		// The second objective is a maximization problem. x_new must be negative
		if (obj == 1) {

			// Normalize a set of continuous values using SoftMax (based on the logistic function)
			for (int i = begin; i < end; ++i) {
				float x_scaled = (pop[i].fitness[obj] - average) / std_deviation;
				float x_new = 1.0f / (1.0f + exp(-x_scaled));
				pop[i].fitness[obj] = -x_new;
			}
		}
		else {

			// Normalize a set of continuous values using SoftMax (based on the logistic function)
			for (int i = begin; i < end; ++i) {
				float x_scaled = (pop[i].fitness[obj] - average) / std_deviation;
				float x_new = 1.0f / (1.0f + exp(-x_scaled));
				pop[i].fitness[obj] = x_new;
			}
		}
	}
	/* -- */
}

/**
 * @brief Evaluation of each individual.
 * @param pop Current population.
 * @param begin The first individual to evaluate.
 * @param end The "end-1" position is the last individual to evaluate.
 * @param dataBase The database which will contain the instances and the features.
 * @param nInstances The number of instances (rows) of the database.
 * @param nFeatures The number of features (columns) of the database.
 * @param nObjectives The number of objectives.
 * @param selInstances The instances chosen as initial centroids.
 */
void cpu_evaluation(individual *pop, const int begin, const int end, const float *const dataBase, const int nInstances, const int nFeatures, const unsigned char nObjectives, const int *const selInstances) {


	/************ Kmeans algorithm ***********/

	// Evaluate all the individuals and get the first and second objective for them
	cpu_kmeans(pop, begin, end, selInstances, dataBase);


	/******************** Fitness normalization *********************/

	int totalInd = end - begin;
	for (unsigned char obj = 0; obj < nObjectives; ++obj) {

		// Fitness vector average
		float average = 0;
		for (int i = begin; i < end; ++i) {
			average += pop[i].fitness[obj];
		}

		average /= totalInd;

		// Fitness vector variance
		float variance = 0;
		for (int i = begin; i < end; ++i) {
			variance += (pop[i].fitness[obj] - average) * (pop[i].fitness[obj] - average);
		}
		variance /= (totalInd - 1);

		// Fitness vector standard deviation
		float std_deviation = sqrt(variance);

		// The second objective is a maximization problem. x_new must be negative
		if (obj == 1) {

			// Normalize a set of continuous values using SoftMax (based on the logistic function)
			for (int i = begin; i < end; ++i) {
				float x_scaled = (pop[i].fitness[obj] - average) / std_deviation;
				float x_new = 1.0f / (1.0f + exp(-x_scaled));
				pop[i].fitness[obj] = -x_new;
			}
		}
		else {

			// Normalize a set of continuous values using SoftMax (based on the logistic function)
			for (int i = begin; i < end; ++i) {
				float x_scaled = (pop[i].fitness[obj] - average) / std_deviation;
				float x_new = 1.0f / (1.0f + exp(-x_scaled));
				pop[i].fitness[obj] = x_new;
			}
		}
	}
	/* -- */
}

/**
 * @brief Evaluation of each individual.
 * @param pop Current population.
 * @param begin The first individual to evaluate.
 * @param end The "end-1" position is the last individual to evaluate.
 * @param dataBase The database which will contain the instances and the features.
 * @param nInstances The number of instances (rows) of the database.
 * @param nFeatures The number of features (columns) of the database.
 * @param nObjectives The number of objectives.
 * @param selInstances The instances chosen as initial centroids.
 */
void CUDA_evaluation(individual *pop, const int begin, const int end, const float *const dataBase, const int nInstances, const int nFeatures, const unsigned char nObjectives, const int *const selInstances) {


	/************ Kmeans algorithm ***********/

	// Evaluate all the individuals and get the first and second objective for them
	CUDA_kmeans(pop, begin, end, selInstances, dataBase);


	/******************** Fitness normalization *********************/

	int totalInd = end - begin;
	for (unsigned char obj = 0; obj < nObjectives; ++obj) {

		// Fitness vector average
		float average = 0;
		for (int i = begin; i < end; ++i) {
			average += pop[i].fitness[obj];
		}

		average /= totalInd;

		// Fitness vector variance
		float variance = 0;
		for (int i = begin; i < end; ++i) {
			variance += (pop[i].fitness[obj] - average) * (pop[i].fitness[obj] - average);
		}
		variance /= (totalInd - 1);

		// Fitness vector standard deviation
		float std_deviation = sqrt(variance);

		// The second objective is a maximization problem. x_new must be negative
		if (obj == 1) {

			// Normalize a set of continuous values using SoftMax (based on the logistic function)
			for (int i = begin; i < end; ++i) {
				float x_scaled = (pop[i].fitness[obj] - average) / std_deviation;
				float x_new = 1.0f / (1.0f + exp(-x_scaled));
				pop[i].fitness[obj] = -x_new;
			}
		}
		else {

			// Normalize a set of continuous values using SoftMax (based on the logistic function)
			for (int i = begin; i < end; ++i) {
				float x_scaled = (pop[i].fitness[obj] - average) / std_deviation;
				float x_new = 1.0f / (1.0f + exp(-x_scaled));
				pop[i].fitness[obj] = x_new;
			}
		}
	}
	/* -- */
}


/**
 * @brief Gets the hypervolume measure of the population.
 * @param pop Current population.
 * @param nIndFront0 The number of individuals in the front 0.
 * @param nObjectives The number of objectives.
 * @param referencePoint The necessary reference point for calculation.
 * @return The value of the hypervolume.
 */
float getHypervolume(const individual *const pop, const int nIndFront0, const unsigned char nObjectives, const double *const referencePoint) {

	// Generation the points for the calculation of the hypervolume
	double *points = new double[nObjectives * nIndFront0];
	for (int i = 0; i < nIndFront0; ++i) {
		for (unsigned char obj = 0; obj < nObjectives; ++obj) {
			points[(i * nObjectives) + obj] = pop[i].fitness[obj];
		}
	}

	float hypervolume = fpli_hv(points, nObjectives, nIndFront0, referencePoint);
	delete[] points;

	return hypervolume;
}

/**
 * @brief CUDA Kernel used to generate a random number per thread
 * @param d_out The random numbers generated
 * @param max The upper limit of the generation interval
 * @param min The bottom limit of the generation interval
 */
__global__ void cudaRand(int *d_out, const int max, const int min)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState state;
    hiprand_init((unsigned long long)clock() + i, 0, 0, &state);

    float my_rand_float = hiprand_uniform_double(&state);
    my_rand_float *= (max - min+0.99999);
    my_rand_float += min;
    int my_rand = (int)truncf(my_rand_float);

//    assert(my_rand <= max);
//    assert(my_rand >= min);

    d_out[i] = my_rand;
}


/**
 * @brief Gets the initial centroids (instances chosen randomly).
 * @param selInstances Where the instances chosen as initial centroids will be stored.
 * @param nInstances The number of instances (rows) of the database.
 */
void getCentroids(int *selInstances, const int nInstances) {
	// The init centroids will be instances chosen randomly (Forgy's Method)
	for (int k = 0; k < KMEANS; ++k) {
		bool exists = false;
		bool alternate = false;
		int randomInstance;

		/* -- */
		// Avoid repeat centroids
		do {
			randomInstance = rand() % nInstances;
			if(alternate){randomInstance++;if(randomInstance==nInstances){randomInstance=randomInstance-2;}}
			exists = false;

			// Look if the generated index already exists
			for (int kk = 0; kk < k && !exists; ++kk) {
				exists = (randomInstance == selInstances[kk]);
				if(alternate){
					alternate = false;
				}else{
					alternate = true;
				}
			}
		} while (exists);
		/* -- */

		selInstances[k] = randomInstance;
	}
}


/**
 * @brief Generates gnuplot code for data display.
 * @param dataName The name of the file which will contain the fitness of the individuals in the first Pareto front.
 * @param plotName The name of the file which will contain the gnuplot code for data display.
 * @param imageName The name of the file which will contain the image with the data (graphic).
 * @param pop Current population.
 * @param nIndFront0 The number of individuals in the front 0.
 * @param nObjectives The number of objectives.
 * @param referencePoint The reference point used for the hypervolume calculation.
 */
void generateGnuplot(const char *dataName, const char *plotName, const char *imageName, const individual *const pop, const int nIndFront0, const unsigned char nObjectives, const double *const referencePoint) {

	// Open the data file
	FILE *f_data;
	f_data = fopen(dataName, "w");
	if (!f_data) {
		fprintf(stderr, "Error: An error ocurred opening or writting the data file\n");
		exit(-1);
	}

	// Write the data
	fprintf(f_data, "#Objective0");
	for (unsigned char obj = 1; obj < nObjectives; ++obj) {
		fprintf(f_data, "\tObjective%d", obj);
	}
	for (int i = 0; i < nIndFront0; ++i) {
		fprintf(f_data, "\n%f", pop[i].fitness[0]);
		for (unsigned char obj = 1; obj < nObjectives; ++obj) {
			fprintf(f_data, "\t%f", pop[i].fitness[obj]);
		}
	}
	fclose(f_data);

	// Gnuplot is only available for two objectives
	if (nObjectives == 2) {

		// Open the gnuplot script file
		FILE *f_plot;
		f_plot = fopen(plotName, "w");
		if (!f_data) {
			fprintf(stderr, "Error: An error ocurred opening or writting the plot file\n");
			exit(-1);
		}

		// Write the code
		fprintf(f_plot, "#!/usr/bin/gnuplot\n");
		fprintf(f_plot, "set terminal png size 1024,600\n");
		fprintf(f_plot, "set output '%s.png'\n", imageName);
		fprintf(f_plot, "set multiplot\n");
		fprintf(f_plot, "set xlabel \"Objective 0\"\n");
		fprintf(f_plot, "set grid\n");
		fprintf(f_plot, "set title \"Pareto front\"\n");
		fprintf(f_plot, "set ylabel \"Objective 1\"\n");
		fprintf(f_plot, "set size 0.9,0.9\n");
		fprintf(f_plot, "set origin 0.00,0.05\n");
		fprintf(f_plot, "set key center top\n");
		fprintf(f_plot, "plot [0:1][-1:1] '< sort %s' using 1:%d title \"Front 0\" with lp,\\\n", dataName, nObjectives);
		fprintf(f_plot, "\t\"<echo '%f %f'\" title \"Reference point\" with points,\\\n", referencePoint[0], referencePoint[1]);
		fprintf(f_plot, "\t0 title \"Top pareto limit\" with lp;\n");
		fprintf(f_plot, "set nomultiplot\n");
		fprintf(f_plot, "reset\n");
		fclose(f_plot);
	}
	else {
		fprintf(stdout, "Gnuplot is only available for two objectives. Not generated gnuplot file\n");
	}
}
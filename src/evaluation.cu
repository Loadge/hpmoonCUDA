#include "hip/hip_runtime.h"
/**
 * @file evaluation.cpp
 * @author Juan José Escobar Pérez
 * @date 20/06/2015
 * @brief File with the necessary implementation for the evaluation of the individuals
 *
 */

/********************************** Includes **********************************/

#include "evaluation.h"
#include "hv.h"
#include <stdio.h> // fprintf...
#include <stdlib.h> // malloc, rand...
#include <string.h> // memset...
#include <math.h> // sqrt, INFINITY...

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

/********************************* Methods ********************************/

static inline int nextPowerOfTwo(int n) {
    n--;

    n = n >>  1 | n;
    n = n >>  2 | n;
    n = n >>  4 | n;
    n = n >>  8 | n;
    n = n >> 16 | n;
//  n = n >> 32 | n;    //  For 64-bit ints

    return ++n;
}

/* -- * /
__global__ static
void cuda_WCSSICSS( 
						float *centroids,					//OK
						unsigned char *member_chromosome,	//OK
						float * distCentroids,				//OK
						bool * mapping,						//OK
						float * result_sumWithin,
						float * result_sumInter
					)
{
	const int d_totalDistances = KMEANS * N_INSTANCES;
	int tx = threadIdx.x;

	__shared__ float sharedCentroids [KMEANS * N_FEATURES];
	__shared__ unsigned char sharedMember_chromosome [N_FEATURES];		
	__shared__ float sharedDistCentroids[d_totalDistances];
	__shared__ bool  sharedMapping 		[d_totalDistances];
	__shared__ float sharedResult_sumWithin;
	__shared__ float sharedResult_sumInter;

	
	
	//Copiar valores a la memoria compartida;
	if(tx < KMEANS * N_FEATURES){
		sharedCentroids[tx] = centroids[tx];
	}
	__syncthreads();
	if(tx < N_FEATURES){
		sharedMember_chromosome[tx] = member_chromosome[tx];
	}
	__syncthreads();
	if(tx < d_totalDistances){
		sharedDistCentroids[tx] = distCentroids[tx];
	}
	__syncthreads();
	if(tx < d_totalDistances){
		sharedMapping[tx] = mapping[tx];
	}

	__shared__ int totalCoord;
	__shared__ float sumWithin[KMEANS];
	__shared__ float sumInter[KMEANS];
//		for (int k = 0; k < KMEANS; ++k) {
	__syncthreads();


	if(tx == 0){
		totalCoord = KMEANS * N_FEATURES;
	}

	if(tx < KMEANS){
		int posCentroids = tx * N_FEATURES;
		int posDistCentr = tx * N_INSTANCES;

//		sumWithin[tx] = 0.0f;
//		sumInter[tx] = 0.0f;

		// Within-cluster
		for (int i = 0; i < N_INSTANCES; ++i) {
			if (sharedMapping[posDistCentr + i]) {
				sharedResult_sumWithin = sharedResult_sumWithin + sharedDistCentroids[posDistCentr + i];
			}
		}

		// Inter-cluster
		for (int i = posCentroids + N_FEATURES; i < totalCoord; i += N_FEATURES) {
			float sum = 0.0f;
			for (int f = 0; f < N_FEATURES; ++f) {
				if (sharedMember_chromosome[f] & 1) {
					sum += (sharedCentroids[posCentroids + f] - sharedCentroids[i + f]) * (sharedCentroids[posCentroids + f] - sharedCentroids[i + f]);
				}
			}
			__syncthreads();
			sharedResult_sumInter += sqrt(sum);
			__syncthreads();
		}
	}//WCSS and ICSS minimization process
/* -- * /	
	__syncthreads();
//	if(tx < KMEANS){						//TODO: make a proper reduction operation
	if(tx == 0){
		sharedResult_sumWithin = 0;
		sharedResult_sumInter = 0;
		for(int i=0; i < KMEANS; i++){
			sharedResult_sumWithin += sumWithin[tx];
			sharedResult_sumInter  += sumInter[tx];
		}
	}
	__syncthreads();
/* -- * /	
	if(tx==0){
//		*(result_sumWithin) = 5.0;
//		*(result_sumInter) = 5.0;
		*(result_sumWithin) = sharedResult_sumWithin;
		*(result_sumInter) = sharedResult_sumInter;

	}
/* -- * /


}
/* -- */




__global__ static
void cuda_Convergence(	float *dataBase, 
						float *centroids,
						unsigned char *member_chromosome,	
						float * distCentroids,				
						bool * mapping,						
						bool * newMapping,					
						bool * auxMapping,
						int *samples_in_k,  		
						bool * converged,
						bool * convergedThread,
						int * nvueltas
					)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = threadIdx.x;
	//int tx = threadIdx.x;
	const int d_totalDistances = KMEANS * N_INSTANCES;

	//extern __shared__ int sharedMemory[]; //main OPT point
	__shared__ float sharedCentroids [KMEANS * N_FEATURES];
	__shared__ unsigned char sharedMember_chromosome [N_FEATURES];
	__shared__ float sharedDistCentroids[d_totalDistances];
	__shared__ bool  sharedMapping 		[d_totalDistances];
	__shared__ bool  sharedNewMapping 	[d_totalDistances];
	__shared__ int sharedSamples_in_k [KMEANS];
	__shared__ int SharedThreadLater[N_INSTANCES];


	//Copiar valores a la memoria compartida;
	if(idx < KMEANS * N_FEATURES){
		sharedCentroids[idx] = centroids[idx];
	}
	__syncthreads();
	if(idx < N_FEATURES){
		sharedMember_chromosome[idx] = member_chromosome[idx];
	}
	__syncthreads();
	if(idx < d_totalDistances){
		sharedDistCentroids[idx] = distCentroids[idx];
	}
	__syncthreads();
	if(idx < d_totalDistances){
		sharedMapping[idx] = mapping[idx];
	}
	__syncthreads();
	if(idx < d_totalDistances){
		sharedNewMapping[idx]=false;
		sharedMapping[idx]=false;
	}
	__syncthreads();
	if(idx==0){
		*(nvueltas)=0;
		*converged= false;
	}
	__syncthreads();
/* -- */
//		for (int maxIter = 0; maxIter < MAX_ITER_KMEANS && (!(*converged==true)); ++maxIter) {
			__syncthreads();
			// The mapping table is cleaned in each iteration
			//Works when number of threads >=  KMEANS * N_INSTANCES
			if(idx < d_totalDistances){
				sharedNewMapping[idx] = false;
			}
			__syncthreads();
			if(idx < KMEANS){
				sharedSamples_in_k[idx] = 0;
			}
			__syncthreads();
			//A single thread executes the necessary work to compute an instance. Future improvements can me made. OPT
			if(idx < N_INSTANCES){
//			for (int i = idx; i < N_INSTANCES; i += blockDim.x) {
/* -- */
				float minDist = INFINITY;
				int selectCentroid = -1;
				int pos = N_FEATURES * idx;

				for (int k = 0; k < KMEANS; ++k) {
					float sum = 0.0f;
					int posCentroids = k * N_FEATURES;
					int posDistCentr = k * N_INSTANCES;
					for (int f = 0; f < N_FEATURES; ++f) {
						if (sharedMember_chromosome[f] & 1) {
							//Multiple accesses to global memory. Better if they were in shared memory. OPT
							sum += (dataBase[pos + f] - sharedCentroids[posCentroids + f]) * (dataBase[pos + f] - sharedCentroids[posCentroids + f]);
						}
					}//f

					float euclidean = sqrt(sum);
					sharedDistCentroids[posDistCentr + idx] = euclidean; //Access to global memory. OPT
					if (euclidean < minDist) {
						minDist = euclidean;
						selectCentroid = k;
					}
				}//k
/* -- */
				__syncthreads();
				SharedThreadLater[idx]= selectCentroid;
//				sharedSamples_in_k[selectCentroid] = atomicAdd(&sharedSamples_in_k[selectCentroid], 1);


				sharedNewMapping[(selectCentroid * N_INSTANCES) + idx] = true;
				
//				sharedSamples_in_k[selectCentroid]++;
				__syncthreads();
			}//i
			__syncthreads();
			if(idx==0){
				for(int i=0; i<N_INSTANCES; i++){
					sharedSamples_in_k[SharedThreadLater[i]]++;
				}
			}
			/* -- */
			// Has the algorithm converged?
			//A number of "KMEANS" threads check if the algorithm converged.
			//They put their result in common in the next section
/* -- */			
			if(idx==0){
				*(converged)=true;
				for (int k = 0; k < KMEANS && ((*converged)==true); ++k) { 
					int posMapping = k * N_INSTANCES;
					for (int i = 0; i < N_INSTANCES && ((*converged)==true); ++i) {
						if (sharedNewMapping[posMapping + i] != sharedMapping[posMapping + i]) {
							*converged = false;
						}
					}
				}
			}
			__syncthreads();
/* -- */
			if (!(*converged==true)) {
				// Update the position of the centroids
				if(idx < KMEANS){
					int posCentroids = idx * N_FEATURES;
					int posMapping = idx * N_INSTANCES;
					for (int f = 0; f < N_FEATURES; ++f) {
						float sum = 0.0f;
						if (sharedMember_chromosome[f] & 1) {
							for (int i = 0; i < N_INSTANCES; ++i) {
								if (sharedNewMapping[posMapping + i]) {
									sum += dataBase[(N_FEATURES * i) + f];
								}
							}
							sharedCentroids[posCentroids + f] = sum / sharedSamples_in_k[idx];
						}
					}
				}
/* -- */
				// Swap mapping tables
				if(idx < d_totalDistances){
					auxMapping[idx]=sharedNewMapping[idx];
					__syncthreads();
					sharedNewMapping[idx]=sharedMapping[idx];
					__syncthreads();
					sharedMapping[idx]=auxMapping[idx];
					__syncthreads();
				}
/* -- */
			}//!converged
/* -- * /
//			if(tx==0){
//				*nvueltas=(*nvueltas)+1;
//			}
//			__syncthreads();
/* -- * /
		}//max-iter
/* -- */
		//Copiar los valores de salida
	if(idx < KMEANS * N_FEATURES){
		centroids[idx] = sharedCentroids[idx];
	}
	if(idx < d_totalDistances){
		distCentroids[idx] = sharedDistCentroids[idx];
	}
/* -- */	
	if(idx < d_totalDistances){
		mapping[idx]	= sharedMapping[idx];
	}
/* -- */
	if(idx < d_totalDistances){
		newMapping[idx] = sharedNewMapping[idx];
	}
/* -- */
	if(idx < KMEANS){
		samples_in_k[idx] = sharedSamples_in_k[idx];
	}
/* -- */
}




/**
 * @brief K-means algorithm which minimize the within-cluster and maximize Inter-cluster sum of squares (WCSS and ICSS)
 * @param pop Current population
 * @param begin The first individual to evaluate
 * @param end The "end-1" position is the last individual to evaluate
 * @param selInstances The instances chosen as initial centroids
 * @param dataBase The database which will contain the instances and the features
 */
void kmeans(individual *pop, const int begin, const int end, const int *const selInstances, const float *const dataBase) {

	bool *mapping = (bool*) malloc(KMEANS * N_INSTANCES * sizeof(bool));
	bool *newMapping = (bool*) malloc(KMEANS * N_INSTANCES * sizeof(bool));

	// Evaluate all the individuals
	for (int ind = begin; ind < end; ++ind) {
		float centroids[KMEANS * N_FEATURES];

		// The centroids will have the selected features of the individual
		for (int k = 0; k < KMEANS; ++k) {
			int posDataBase = selInstances[k] * N_FEATURES;
			int posCentr = k * N_FEATURES;

			for (int f = 0; f < N_FEATURES; ++f) {
				if (pop[ind].chromosome[f] & 1) {
					centroids[posCentr + f] = dataBase[posDataBase + f];
				}
			}
		}

		/******************** Convergence process *********************/

		// Initialize the array of minimum distances and the mapping table
		const int totalDistances = KMEANS * N_INSTANCES;
		float distCentroids[KMEANS * N_INSTANCES];
		int samples_in_k[KMEANS];

		// Initialize the mapping table
		for (int i = 0; i < totalDistances; ++i) {
			mapping[i] = false;
		}

		/* ZONA PARALELA*/
/* -- */
		hipSetDevice(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

//        printf("\n%d procesadores!", _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);

		//Allocate device memory-------------------------------------------
		float *d_dataBase;	
		size_t size_1 = N_INSTANCES * N_FEATURES * sizeof(float);			//NO VARÍA
		checkCudaErrors(hipMalloc((void **)&d_dataBase, size_1));

		float *d_centroids;	
		size_t size_2 = KMEANS * N_FEATURES * sizeof(float);
		checkCudaErrors(hipMalloc((void **)&d_centroids, size_2));

		unsigned char *d_member_chromosome;	
		size_t size_3 = N_FEATURES * sizeof(unsigned char);					//NO VARÍA
		checkCudaErrors(hipMalloc((void **)&d_member_chromosome, size_3));

		float * d_distCentroids;	
		size_t size_4 = KMEANS * N_INSTANCES * sizeof(float);
		checkCudaErrors(hipMalloc((void **)&d_distCentroids, size_4));

		bool * d_mapping;		
		size_t size_5 = KMEANS * N_INSTANCES * sizeof(bool);	
		checkCudaErrors(hipMalloc((void **)&d_mapping, size_5));

		bool * d_newMapping;
		checkCudaErrors(hipMalloc((void **)&d_newMapping, size_5));

		bool * d_auxMapping;
		checkCudaErrors(hipMalloc((void **)&d_auxMapping, size_5));		

		int *d_samples_in_k; 
		size_t size_6 = KMEANS * sizeof(int);
		checkCudaErrors(hipMalloc((void **)&d_samples_in_k, size_6));	

		//d_converged is meant for internal use in cuda function
		bool * d_converged;
		checkCudaErrors(hipMalloc(&d_converged, sizeof(bool)));

		int * d_nvueltas;
		checkCudaErrors(hipMalloc(&d_nvueltas, sizeof(int)));

		bool * d_convergedThread;
		size_t size_7 = KMEANS * sizeof(bool);
		checkCudaErrors(hipMalloc((void **)&d_convergedThread, size_7));

		//Capture individual chromosome
		unsigned char h_member_chromosome[N_FEATURES];
		for(int i=0; i<N_FEATURES; i++){
			h_member_chromosome[i] = pop[ind].chromosome[i];
		}

		//Se decide el nº de bloques
//		unsigned int numThreadsPerBlock = nextPowerOfTwo(KMEANS * N_INSTANCES);
		unsigned int numThreadsPerBlock = KMEANS * N_INSTANCES;
//		unsigned int numBlocks = 1+ (numThreadsPerBlock)/ (_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
		unsigned int numBlocks = ((KMEANS * N_INSTANCES)+numThreadsPerBlock-1) / numThreadsPerBlock;
		if(numBlocks > (_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount)){numBlocks = (_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);}	
		long unsigned int BlockSharedDataSize = 
//												N_INSTANCES * N_FEATURES * sizeof(float) +	//database
												KMEANS * N_FEATURES * sizeof(float) +		//centroids
												N_FEATURES * sizeof(unsigned char) +		//member_chromosome
												KMEANS * N_INSTANCES * sizeof(float) +		//distCentroids
												KMEANS * N_INSTANCES * sizeof(bool) +		//mapping
												KMEANS * N_INSTANCES * sizeof(bool) +		//newMapping
//												KMEANS * N_INSTANCES * sizeof(bool) +		//d_auxMapping
//												KMEANS * sizeof(int) +						//samples_in_k
												sizeof(bool)		  +						//converged
												KMEANS * sizeof(bool) +						//convergedThread
												sizeof(int) +								//nvueltas
												0;								

		if (BlockSharedDataSize > deviceProp.sharedMemPerBlock) {printf("WARNING: Your CUDA hardware has insufficient block shared memory.\n");}
/* -- */
		//Copy values in device memory-------------------------------------
		checkCudaErrors(hipMemcpy(d_dataBase, dataBase, size_1, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_centroids, centroids, size_2, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_member_chromosome, h_member_chromosome, size_3, hipMemcpyHostToDevice));
		//checkCudaErrors(hipMemcpy(d_distCentroids, distCentroids, size_4, hipMemcpyHostToDevice));
		//checkCudaErrors(hipMemcpy(d_mapping, mapping, size_5, hipMemcpyHostToDevice));
		//This variables are initialized within the device kernel
		//checkCudaErrors(hipMemcpy(d_newMapping, newMapping, size_5, hipMemcpyHostToDevice));
		//checkCudaErrors(hipMemcpy(d_samples_in_k, samples_in_k, size_6, hipMemcpyHostToDevice));
/* -- */
		

/* -- */
		printf("\nnumThreadsPerBlock: %d", numThreadsPerBlock);
		printf("\nnumBlocks: %d", numBlocks);
		printf("\n Numerajo: %d", (_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount));
		printf("\n Se van a lanzar %d hebras repartidas en %d bloques", numThreadsPerBlock*numBlocks, numBlocks);
		printf("\nUso de memoria compartida: %lu bytes / %lu", BlockSharedDataSize, deviceProp.sharedMemPerBlock);

/* -- */
		int nVueltas=0;
		// To avoid poor performance, up to "MAX_ITER_KMEANS" iterations are executed
		bool converged = false;
//		for (int maxIter = 0; maxIter < MAX_ITER_KMEANS && !converged; ++maxIter) {
			// The mapping table is cleaned in each iteration
			for (int i = 0; i < totalDistances; ++i) {
				newMapping[i] = false;
			}
			for (int i = 0; i < KMEANS; ++i) {
				samples_in_k[i] = 0;
			}
/* -- */
//			printf("\n -----[%d]----- Parte secuencial: calcular distancias Euclídeas", maxIter);
			// Calculate all distances (Euclidean distance) between each instance and the centroids
			for (int i = 0; i < N_INSTANCES; ++i) {
/* -- */
				float minDist = INFINITY;
				int selectCentroid = -1;
				int pos = N_FEATURES * i;
				for (int k = 0; k < KMEANS; ++k) {  //Para cada centroide
					float sum = 0.0f;
					int posCentroids = k * N_FEATURES;		//Nos situamos en el centroide
					int posDistCentr = k * N_INSTANCES;		//Nos situamos en [	i1 i2 i3 i4 i5 i6 i7 ]
					for (int f = 0; f < N_FEATURES; ++f) {	//					centroide 3
						if (pop[ind].chromosome[f] & 1) {
							sum += (dataBase[pos + f] - centroids[posCentroids + f]) * (dataBase[pos + f] - centroids[posCentroids + f]);
						}
					}//f

					float euclidean = sqrt(sum);
					distCentroids[posDistCentr + i] = euclidean;
					if (euclidean < minDist) {
						minDist = euclidean;
						selectCentroid = k;
					}
				}//k

				newMapping[(selectCentroid * N_INSTANCES) + i] = true;
				samples_in_k[selectCentroid]++;
			}//i
/*  -- */
//			printf("\n -----[%d]----- Parte secuencial: ver si ha convergido o no", maxIter);
			// Has the algorithm converged?
			converged = true;
			for (int k = 0; k < KMEANS && converged; ++k) { 
				int posMapping = k * N_INSTANCES;
				for (int i = 0; i < N_INSTANCES && converged; ++i) {
					if (newMapping[posMapping + i] != mapping[posMapping + i]) {
						converged = false;
					}
				}
			}
/* -- */
			if (!converged) {
				// Update the position of the centroids
				for (int k = 0; k < KMEANS; ++k) {
					int posCentroids = k * N_FEATURES;
					int posMapping = k * N_INSTANCES;
					for (int f = 0; f < N_FEATURES; ++f) {
						float sum = 0.0f;
						if (pop[ind].chromosome[f] & 1) {
							for (int i = 0; i < N_INSTANCES; ++i) {
								if (newMapping[posMapping + i]) {
									sum += dataBase[(N_FEATURES * i) + f];
								}
							}
							centroids[posCentroids + f] = sum / samples_in_k[k];
						}
					}
				}
/* -- */
//				printf("\n -----[%d]----- Parte secuencial: intercambiar mappings", maxIter);
				// Swap mapping tables
				bool *aux = newMapping;
				newMapping = mapping;
				mapping = aux;
/* -- */
			}//!converged
//			printf("\n -----[%d]----- Parte secuencial: Finalizó MAXITER", maxIter);	

/* -- * /
			nVueltas++;
		}//maxiter----------------------------------------------------------------------------
/* -- */

		float gpu_distCentroids[KMEANS * N_INSTANCES];
//		bool *gpu_newMapping = 	(bool*) malloc(KMEANS * N_INSTANCES * sizeof(bool));
//		bool *gpu_mapping = 	(bool*) malloc(KMEANS * N_INSTANCES * sizeof(bool));
		bool gpu_newMapping[KMEANS * N_INSTANCES];
		bool gpu_mapping[KMEANS * N_INSTANCES];

		int gpu_nvueltas;
		bool gpu_converged;
		int gpu_samples_in_k[KMEANS];
		float gpu_centroids[KMEANS * N_FEATURES];
		/* ZONA PARALELA*/
		//**********************************************************************************************
//		for (int maxIter = MAX_ITER_KMEANS-1; maxIter < MAX_ITER_KMEANS && !converged; ++maxIter) {

			cuda_Convergence <<< numBlocks, numThreadsPerBlock /*, BlockSharedDataSize */ >>> (	d_dataBase, 
																		d_centroids, 
																		d_member_chromosome, 
																		d_distCentroids, 
																		d_mapping, 
																		d_newMapping,
																		d_auxMapping, 
																		d_samples_in_k,
																		d_converged,
																		d_convergedThread,
																		d_nvueltas
																		);
			hipDeviceSynchronize();

			checkCudaErrors(hipMemcpy(gpu_distCentroids, d_distCentroids, size_4, hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(gpu_newMapping, d_newMapping, size_5, hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(gpu_mapping, d_mapping, size_5, hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(gpu_samples_in_k, d_samples_in_k, size_6, hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(gpu_centroids, d_centroids, size_2, hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(&gpu_converged, d_converged, sizeof(bool), hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(&gpu_nvueltas, d_nvueltas, sizeof(int), hipMemcpyDeviceToHost));

			//checkCudaErrors(hipMemcpy(gpu_converged, d_converged, sizeof(bool), hipMemcpyDeviceToHost));
//			hipMemcpyFromSymbol(&gpu_converged, HIP_SYMBOL("CUDAconverged"), sizeof(bool), 0, hipMemcpyDeviceToHost);
			//**********************************************************************************************
/* -- * /
		}//max iter
			/* -- */
			//Comprobación de si los resultados son iguales que en secuencial:
			printf("\nComprobando si todo está OC\n");

			int tam_mapping = KMEANS * N_INSTANCES;
			for(int i=0; i< KMEANS * N_FEATURES; i++){
				if(gpu_distCentroids[i] != distCentroids[i]){
					printf("\ndistCentroids[%d] no encaja con la versión secuencial.\n", i);
				}else{
					printf("\ndistCentroids[%d] encaja con la versión secuencial.\n", i);
				}
				printf("%f\n%f\n", distCentroids[i], gpu_distCentroids[i]);
			}
/* -- */
			for(int i=0; i<KMEANS; i++){
				if(gpu_samples_in_k[i] != samples_in_k[i]){
					printf("\nsamples_in_k[%d] no encaja con la versión secuencial.\n", i);
					printf("[%d]   %d %d\n", i, samples_in_k[i], gpu_samples_in_k[i]);
				}
			}
/* -- */
			for(int i=0; i< KMEANS * N_FEATURES; i++){
				if(gpu_centroids[i] != centroids[i]){
					printf("\ncentroids[%d] no encaja con la versión secuencial.\n", i);
				}else{
					printf("\ncentroids[%d] encaja con la versión secuencial.\n", i);
				}
				printf("%f\n%f\n", centroids[i], gpu_centroids[i]);
			}
/* -- */
			printf("\nmappings normales y de la gpu!\n");
			for(int i=0; i<tam_mapping; i++){
				if( (mapping[i] != gpu_mapping[i]) 
								||
					(newMapping[i] != gpu_newMapping[i])  
															){
					if(i<10){
						printf("[%d]   %d %d\n      %d %d\n\n", i, mapping[i], gpu_mapping[i], newMapping[i], gpu_newMapping[i]);
					}else if(i<100){
						printf("[%d]   %d %d\n       %d %d\n\n", i, mapping[i], gpu_mapping[i], newMapping[i], gpu_newMapping[i]);
					}else{
						printf("[%d]   %d %d\n        %d %d\n\n", i, mapping[i], gpu_mapping[i], newMapping[i], gpu_newMapping[i]);
					}
				}
				
			}
/* -- */
			printf("\nConverged seq: %d, en gpu: %d\n", converged, gpu_converged);
			printf("\nnVueltas seq: %d, en gpu: %d\n", nVueltas, gpu_nvueltas);
			printf("\nPRUEBA PASADA TT");
/* -- */
		printf("\n ---------- Parte secuencial: Empezando WCSS y ICSS");
		/************ Minimize the within-cluster and maximize Inter-cluster sum of squares (WCSS and ICSS) *************/

/* -- * /
		float * d_result_sumWithin;
		checkCudaErrors(hipMalloc(&d_result_sumWithin, sizeof(float)));

		float * d_result_sumInter;
		checkCudaErrors(hipMalloc(&d_result_sumInter, sizeof(float)));
		
		numThreadsPerBlock = nextPowerOfTwo(KMEANS);
		numBlocks = 1+ (numThreadsPerBlock)/ (_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
		BlockSharedDataSize = 
												N_INSTANCES * N_FEATURES * sizeof(float) +	//database
												KMEANS * N_FEATURES * sizeof(float) +		//centroids
												N_FEATURES * sizeof(unsigned char) +		//member_chromosome
												KMEANS * N_INSTANCES * sizeof(float) +		//distCentroids
												KMEANS * N_INSTANCES * sizeof(bool) +		//mapping
//												KMEANS * N_INSTANCES * sizeof(bool) +		//newMapping
//												KMEANS * N_INSTANCES * sizeof(bool) +		//d_auxMapping
												KMEANS * sizeof(int) +						//samples_in_k
//												sizeof(bool)		  +						//converged
//												KMEANS * sizeof(bool) +					//convergedThread
//												sizeof(int) +								//result_sumWithin
//												sizeof(int);								//result_sumInter
		if (BlockSharedDataSize > deviceProp.sharedMemPerBlock) {printf("WARNING: Your CUDA hardware has insufficient block shared memory.\n");}
/* -- */
		int totalCoord = KMEANS * N_FEATURES;
		float sumWithin = 0.0f;
		float sumInter = 0.0f;
//		for(int i=0; i < totalCoord; i++){
//			sumasParciales[i]=0;
//		}

		for (int k = 0; k < KMEANS; ++k) {
//			printf("\n\nValor de k: %d", k);
			int posCentroids = k * N_FEATURES;
			int posDistCentr = k * N_INSTANCES;

			// Within-cluster
			for (int i = 0; i < N_INSTANCES; ++i) {
				if (mapping[posDistCentr + i]) {
					sumWithin += distCentroids[posDistCentr + i];
				}
			}

			// Inter-cluster
			for (int i = posCentroids + N_FEATURES; i < totalCoord; i += N_FEATURES) {
//				printf("\nValor de i: %d", i);
				float sum = 0.0f;
				for (int f = 0; f < N_FEATURES; ++f) {
					if (pop[ind].chromosome[f] & 1) {
						sum += (centroids[posCentroids + f] - centroids[i + f]) * (centroids[posCentroids + f] - centroids[i + f]);
					}
				}
				sumInter += sqrt(sum);
			}
		}//WCSS and ICSS minimization process

/* -- * /
		printf("\nSumas parciales:");
		for(int i=0; i < totalCoord; i++){
			printf("\n%f", sumasParciales[i]);
		}
/* -- */
		// First objective function (Within-cluster sum of squares (WCSS))
		pop[ind].fitness[0] = sumWithin;

		// Second objective function (Inter-cluster sum of squares (ICSS))
		pop[ind].fitness[1] = sumInter;

		// Third objective function (Number of selected features)
		//pop[ind].fitness[2] = (float) nSelFeatures;
/* -- */
		printf("\n ---------- Parte secuencial: Terminado WCSS y ICSS");
/* -- * /
		cuda_WCSSICSS <<< numBlocks, numThreadsPerBlock /*, BlockSharedDataSize * / >>> (	
																		d_centroids, 
																		d_member_chromosome, 
																		d_distCentroids, 
																		d_mapping,
																		d_result_sumWithin,
																		d_result_sumInter
																		);

		printf("\n Comprobando si todo está OC con WCSS y ICSS");
		float gpu_result_sumWithin;
		float gpu_result_sumInter;
		bool gpu_mapping_2[KMEANS * N_INSTANCES];
		float gpu_centroids_2[KMEANS * N_FEATURES];
		checkCudaErrors(hipMemcpy(&gpu_result_sumWithin, d_result_sumWithin, sizeof(int), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(&gpu_result_sumInter, d_result_sumInter, sizeof(int), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(gpu_mapping_2, d_mapping, size_5, hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(gpu_centroids_2, d_centroids, size_2, hipMemcpyDeviceToHost));
		
/* -- * /
		for(int i=0; i<tam_mapping; i++){
			printf("\nmappings de la gpu antes y después!\n");
			printf("%d           %d", gpu_mapping[i], gpu_mapping_2[i]);
		}
/* -- * /
		for(int i=0; i< KMEANS * N_FEATURES; i++){
			if(gpu_centroids_2[i] != centroids[i]){
				printf("\ncentroids[%d] no encaja con la versión secuencial.\n", i);
			}else{
				printf("\ncentroids[%d] encaja con la versión secuencial.\n", i);
			}
			printf("%f\n%f\n", centroids[i], gpu_centroids_2[i]);
		}
/* -- * /

		printf("\nsumWithin seq: %f, en gpu: %f\n", sumWithin, gpu_result_sumWithin);
		printf("\nsumInter seq: %f, en gpu: %f\n", sumInter, gpu_result_sumInter);
/* -- */

		checkCudaErrors(hipFree(d_dataBase));  
		checkCudaErrors(hipFree(d_centroids));
		checkCudaErrors(hipFree(d_member_chromosome));
		checkCudaErrors(hipFree(d_distCentroids));
		checkCudaErrors(hipFree(d_mapping));
		checkCudaErrors(hipFree(d_newMapping));
		checkCudaErrors(hipFree(d_auxMapping));
		checkCudaErrors(hipFree(d_samples_in_k));
//		checkCudaErrors(hipFree(d_converged));
		checkCudaErrors(hipFree(d_convergedThread));
		checkCudaErrors(hipFree(d_nvueltas));
//		checkCudaErrors(hipFree(d_result_sumWithin));
//		checkCudaErrors(hipFree(d_result_sumInter));
	}//for each individual
	// Resources used are released
	free(mapping);
	free(newMapping);

}//kmeans




/**
 * @brief Evaluation of each individual
 * @param pop Current population
 * @param begin The first individual to evaluate
 * @param end The "end-1" position is the last individual to evaluate
 * @param dataBase The database which will contain the instances and the features
 * @param nInstances The number of instances (rows) of the database
 * @param nFeatures The number of features (columns) of the database
 * @param nObjectives The number of objectives
 * @param selInstances The instances chosen as initial centroids
 */
void evaluation(individual *pop, const int begin, const int end, const float *const dataBase, const int nInstances, const int nFeatures, const unsigned char nObjectives, const int *const selInstances) {


	/************ Kmeans algorithm ***********/

	// Evaluate all the individuals and get the first and second objective for them
	kmeans(pop, begin, end, selInstances, dataBase);


	/******************** Fitness normalization *********************/

	int totalInd = end - begin;
	for (unsigned char obj = 0; obj < nObjectives; ++obj) {

		// Fitness vector average
		float average = 0;
		for (int i = begin; i < end; ++i) {
			average += pop[i].fitness[obj];
		}

		average /= totalInd;

		// Fitness vector variance
		float variance = 0;
		for (int i = begin; i < end; ++i) {
			variance += (pop[i].fitness[obj] - average) * (pop[i].fitness[obj] - average);
		}
		variance /= (totalInd - 1);

		// Fitness vector standard deviation
		float std_deviation = sqrt(variance);

		// The second objective is a maximization problem. x_new must be negative
		if (obj == 1) {

			// Normalize a set of continuous values using SoftMax (based on the logistic function)
			for (int i = begin; i < end; ++i) {
				float x_scaled = (pop[i].fitness[obj] - average) / std_deviation;
				float x_new = 1.0f / (1.0f + exp(-x_scaled));
				pop[i].fitness[obj] = -x_new;
			}
		}
		else {

			// Normalize a set of continuous values using SoftMax (based on the logistic function)
			for (int i = begin; i < end; ++i) {
				float x_scaled = (pop[i].fitness[obj] - average) / std_deviation;
				float x_new = 1.0f / (1.0f + exp(-x_scaled));
				pop[i].fitness[obj] = x_new;
			}
		}
	}
	/* -- */
}


/**
 * @brief Gets the hypervolume measure of the population
 * @param pop Current population
 * @param nIndFront0 The number of individuals in the front 0
 * @param nObjectives The number of objectives
 * @param referencePoint The necessary reference point for calculation
 * @return The value of the hypervolume
 */
float getHypervolume(const individual *const pop, const int nIndFront0, const unsigned char nObjectives, const double *const referencePoint) {

	// Generation the points for the calculation of the hypervolume
	double *points = new double[nObjectives * nIndFront0];
	for (int i = 0; i < nIndFront0; ++i) {
		for (unsigned char obj = 0; obj < nObjectives; ++obj) {
			points[(i * nObjectives) + obj] = pop[i].fitness[obj];
		}
	}

	float hypervolume = fpli_hv(points, nObjectives, nIndFront0, referencePoint);
	delete[] points;

	return hypervolume;
}


__global__ void cudaRand(int *d_out, const int max, const int min)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState state;
    hiprand_init((unsigned long long)clock() + i, 0, 0, &state);

    float my_rand_float = hiprand_uniform_double(&state);
    my_rand_float *= (max - min+0.99999);
    my_rand_float += min;
    int my_rand = (int)truncf(my_rand_float);

//    assert(my_rand <= max);
//    assert(my_rand >= min);

    d_out[i] = my_rand;
}


/**
 * @brief Gets the initial centroids (instances chosen randomly)
 * @param selInstances Where the instances chosen as initial centroids will be stored
 * @param nInstances The number of instances (rows) of the database
 */
void getCentroids(int *selInstances, const int nInstances) {

/* -- * /
										 
										//OPTIMIZAR <<<<<---- El número de bloques
										//ARREGLAR  <<<<<---- numeros no repetidos
	int * h_v = new int[KMEANS];
	int * d_out;
	//1D grid of 1D blocks
	hipMalloc((void**)&d_out, KMEANS*sizeof(int));
	cudaRand<<<1, KMEANS >>> (d_out, 0, KMEANS);
	hipMemcpy(h_v, d_out, KMEANS * sizeof(int), hipMemcpyDeviceToHost);
	

	for(size_t i=0; i<KMEANS;i++){
		printf("%d ", h_v[i]);
	}
	selInstances = h_v;
	
	printf("\n");
	hipFree(d_out);
	delete[] h_v;
/* -- */
	// The init centroids will be instances chosen randomly (Forgy's Method)
	for (int k = 0; k < KMEANS; ++k) {
		bool exists = false;
		bool alternate = false;
		int randomInstance;

		/* -- */
		// Avoid repeat centroids
		do {
			randomInstance = rand() % nInstances;
			if(alternate){randomInstance++;if(randomInstance==nInstances){randomInstance=randomInstance-2;}}
			exists = false;

			// Look if the generated index already exists
			for (int kk = 0; kk < k && !exists; ++kk) {
				exists = (randomInstance == selInstances[kk]);
				if(alternate){
					alternate = false;
				}else{
					alternate = true;
				}
			}
		} while (exists);
		/* -- */

		selInstances[k] = randomInstance;
	}
	printf("\nCentroides: ");
	for(int i=0;i<KMEANS;i++){
		printf("%d ", selInstances[i]);
	}
	printf("\n");
}


/**
 * @brief Generates gnuplot code for data display
 * @param dataName The name of the file which will contain the fitness of the individuals in the first Pareto front
 * @param plotName The name of the file which will contain the gnuplot code for data display
 * @param imageName The name of the file which will contain the image with the data (graphic)
 * @param pop Current population
 * @param nIndFront0 The number of individuals in the front 0
 * @param nObjectives The number of objectives
 * @param referencePoint The reference point used for the hypervolume calculation
 */
void generateGnuplot(const char *dataName, const char *plotName, const char *imageName, const individual *const pop, const int nIndFront0, const unsigned char nObjectives, const double *const referencePoint) {

	// Open the data file
	FILE *f_data;
	f_data = fopen(dataName, "w");
	if (!f_data) {
		fprintf(stderr, "Error: An error ocurred opening or writting the data file\n");
		exit(-1);
	}

	// Write the data
	fprintf(f_data, "#Objective0");
	for (unsigned char obj = 1; obj < nObjectives; ++obj) {
		fprintf(f_data, "\tObjective%d", obj);
	}
	for (int i = 0; i < nIndFront0; ++i) {
		fprintf(f_data, "\n%f", pop[i].fitness[0]);
		for (unsigned char obj = 1; obj < nObjectives; ++obj) {
			fprintf(f_data, "\t%f", pop[i].fitness[obj]);
		}
	}
	fclose(f_data);

	// Gnuplot is only available for two objectives
	if (nObjectives == 2) {

		// Open the gnuplot script file
		FILE *f_plot;
		f_plot = fopen(plotName, "w");
		if (!f_data) {
			fprintf(stderr, "Error: An error ocurred opening or writting the plot file\n");
			exit(-1);
		}

		// Write the code
		fprintf(f_plot, "#!/usr/bin/gnuplot\n");
		fprintf(f_plot, "set terminal png size 1024,600\n");
		fprintf(f_plot, "set output '%s.png'\n", imageName);
		fprintf(f_plot, "set multiplot\n");
		fprintf(f_plot, "set xlabel \"Objective 0\"\n");
		fprintf(f_plot, "set grid\n");
		fprintf(f_plot, "set title \"Pareto front\"\n");
		fprintf(f_plot, "set ylabel \"Objective 1\"\n");
		fprintf(f_plot, "set size 0.9,0.9\n");
		fprintf(f_plot, "set origin 0.00,0.05\n");
		fprintf(f_plot, "set key center top\n");
		fprintf(f_plot, "plot [0:1][-1:1] '< sort %s' using 1:%d title \"Front 0\" with lp,\\\n", dataName, nObjectives);
		fprintf(f_plot, "\t\"<echo '%f %f'\" title \"Reference point\" with points,\\\n", referencePoint[0], referencePoint[1]);
		fprintf(f_plot, "\t0 title \"Top pareto limit\" with lp;\n");
		fprintf(f_plot, "set nomultiplot\n");
		fprintf(f_plot, "reset\n");
		fclose(f_plot);
	}
	else {
		fprintf(stdout, "Gnuplot is only available for two objectives. Not generated gnuplot file\n");
	}
}